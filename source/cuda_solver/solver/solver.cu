#include "hip/hip_runtime.h"
#include <omp.h>

// Include Cuda Kernel headers
#include "cuda/typedef.cuh"
#include "kernel/kernel_compute.cuh"
#include "system/system_parameters.hpp"
#include "cuda/cuda_matrix.cuh"
#include "solver/solver.hpp"
#include "misc/commandline_io.hpp"

namespace PHOENIX {

/*
* Helper variable for caching the current time for FFT evaluations.
* We dont need this variable anywhere else, so we just create it
* locally to this file here.
*/
// TODO: Make these member variables of the solver class
PHOENIX::Type::real fft_cached_t = 0.0;
bool first_time = true;

bool Solver::iterate( bool force_fixed_time_step ) {
    if ( system.use_adaptive_timestep && system.use_adaptive_timestep != is_adaptive_ ) {
        std::cout << CLIO::prettyPrint( "Cannot use variable time step with this solver.", CLIO::Control::Error ) << std::endl;
        system.use_adaptive_timestep = false;
    }

    // Check if the maximum time has been reached
#ifndef BENCH
    if ( system.p.t >= system.t_max )
        return false;
#endif

    // If required, calculate new set of random numbers.
    // TODO: move this back into subgrids, because for large number of subgrids this will look very correlated!
    if ( system.evaluateStochastic() ) {
        auto args = generateKernelArguments();
        auto [block_size, grid_size] = getLaunchParameters( 1, system.p.subgrid_N2_with_halo );
        if ( first_time ) {
            first_time = false;
            CALL_FULL_KERNEL( Kernel::initialize_random_number_generator, "random_number_init", grid_size, block_size, 0, system.random_seed, args.dev_ptrs.random_state, system.p.subgrid_N2_with_halo );
            std::cout << CLIO::prettyPrint( "Initialized Random Number Generator", CLIO::Control::Info ) << std::endl;
        }
        CALL_FULL_KERNEL( Kernel::generate_random_numbers, "random_number_gen", grid_size, block_size, 0, args.dev_ptrs.random_state, args.dev_ptrs.random_number, system.p.subgrid_N2_with_halo, system.p.stochastic_amplitude * std::sqrt( system.p.dt ), system.p.stochastic_amplitude * std::sqrt( system.p.dt ) );
    }

    updateKernelTime();

    // Increase t.
    system.p.t = system.p.t + system.p.dt;

    // Iterate the solver
    step( system.use_adaptive_timestep && !force_fixed_time_step );
    // Call the normalization for imaginary time propagation if required
    if ( system.imag_time_amplitude != 0.0 )
        normalizeImaginaryTimePropagation();

    // For statistical purposes, increase the iteration counter
    system.iteration++;

    // FFT Guard
    if ( system.p.t - fft_cached_t < system.fft_every )
        return true;

    // Calculate the FFT
    fft_cached_t = system.p.t;
    applyFFTFilter( system.fft_mask.size() > 0 );

    return true;
}

bool Solver::adaptTimeStep( const Type::real power, bool use_discrete_update_steps ) {
    auto msum = matrix.buffer_wavefunction_plus.transformReduce( Type::complex( 0.0 ), CUDAMatrix<Type::complex>::transform_abs2(), CUDAMatrix<Type::complex>::transform_sum() );
    Type::real normalization_factor = CUDA::real( msum );
    Type::real integrated_error = matrix.rk_error.sum();

    Type::real final_error = std::abs( integrated_error / normalization_factor );
    Type::real dh_arg = system.tolerance / 2.0 / CUDA::max( std::numeric_limits<Type::real>::min(), final_error );
    Type::real dh = std::pow<Type::real>( dh_arg, Type::real( power ) );

    if ( std::isnan( dh ) ) {
        dh = 0.9;
        final_error = std::numeric_limits<Type::real>::max();
    }
    if ( std::isnan( final_error ) ) {
        dh = 0.9;
        final_error = std::numeric_limits<Type::real>::max();
    }

    //  Set new timestep
    Type::real new_dt = std::min( system.p.dt * dh, system.dt_max );
    if ( new_dt < system.dt_min ) {
        new_dt = system.dt_min;
    }
    system.p.dt = new_dt;
    
    updateKernelTime();

    if (final_error < system.tolerance) {
        // Accept the solution and set the new time step
        return true;
    }
    
    return false;
}

void Solver::swapBuffers() {
    matrix.wavefunction_plus.swap( matrix.buffer_wavefunction_plus );
    matrix.reservoir_plus.swap( matrix.buffer_reservoir_plus );
    if ( system.use_twin_mode ) {
        matrix.wavefunction_minus.swap( matrix.buffer_wavefunction_minus );
        matrix.reservoir_minus.swap( matrix.buffer_reservoir_minus );
    }
}

} // namespace PHOENIX