#include "hip/hip_runtime.h"
#include <omp.h>

// Include Cuda Kernel headers
#include "cuda/typedef.cuh"
#include "kernel/kernel_compute.cuh"
#include "kernel/kernel_summation.cuh"
#include "kernel/kernel_halo.cuh"
#include "system/system_parameters.hpp"
#include "cuda/cuda_matrix.cuh"
#include "solver/gpu_solver.hpp"
#include "misc/commandline_io.hpp"

void PHOENIX::Solver::iterateFixedTimestepCashKarp() {
    SOLVER_SEQUENCE( true /*Capture CUDA Graph*/,

                     CALCULATE_K( 1, Type::real( 0.0 ), wavefunction, reservoir );

                     INTERMEDIATE_SUM_K( 1, Type::real( 1.0 / 5.0 ) );

                     CALCULATE_K( 2, Type::real( 1.0 / 5.0 ), buffer_wavefunction, buffer_reservoir );

                     INTERMEDIATE_SUM_K( 2, Type::real( 3.0 / 40.0 ), Type::real( 9.0 / 40.0 ) );

                     CALCULATE_K( 3, Type::real( 3.0 / 10.0 ), buffer_wavefunction, buffer_reservoir );

                     INTERMEDIATE_SUM_K( 3, Type::real( 3.0 / 1.0 ), Type::real( -9.0 / 10.0 ), Type::real( 6.0 / 5.0 ) );

                     CALCULATE_K( 4, Type::real( 3.0 / 5.0 ), buffer_wavefunction, buffer_reservoir );

                     INTERMEDIATE_SUM_K( 4, Type::real( -11.0 / 54.0 ), Type::real( 5.0 / 2.0 ), Type::real( -70.0 / 27.0 ), Type::real( 35.0 / 27.0 ) );

                     CALCULATE_K( 5, Type::real( 1.0 ), buffer_wavefunction, buffer_reservoir );

                     INTERMEDIATE_SUM_K( 5, Type::real( 1631.0 / 55296.0 ), Type::real( 175.0 / 512.0 ), Type::real( 575.0 / 13824.0 ), Type::real( 44275.0 / 110592.0 ), Type::real( 253.0 / 4096.0 ) );

                     CALCULATE_K( 6, Type::real( 7.0 / 8.0 ), buffer_wavefunction, buffer_reservoir );

                     FINAL_SUM_K( 6, Type::real( 37.0 / 378.0 ), Type::real( 0.0 ), Type::real( 250.0 / 621.0 ), Type::real( 125.0 / 594.0 ), Type::real( 0.0 ), Type::real( 512.0 / 1771.0 ) );

    );
}

void PHOENIX::Solver::iterateVariableTimestepCashKarp() {
    bool accept = false;
    do {
        SOLVER_SEQUENCE( false /*Capture CUDA Graph*/,

                         CALCULATE_K( 1, Type::real( 0.0 ), wavefunction, reservoir );

                         INTERMEDIATE_SUM_K( 1, Type::real( 1.0 / 5.0 ) );

                         CALCULATE_K( 2, Type::real( 1.0 / 5.0 ), buffer_wavefunction, buffer_reservoir );

                         INTERMEDIATE_SUM_K( 2, Type::real( 3.0 / 40.0 ), Type::real( 9.0 / 40.0 ) );

                         CALCULATE_K( 3, Type::real( 3.0 / 10.0 ), buffer_wavefunction, buffer_reservoir );

                         INTERMEDIATE_SUM_K( 3, Type::real( 3.0 / 1.0 ), Type::real( -9.0 / 10.0 ), Type::real( 6.0 / 5.0 ) );

                         CALCULATE_K( 4, Type::real( 3.0 / 5.0 ), buffer_wavefunction, buffer_reservoir );

                         INTERMEDIATE_SUM_K( 4, Type::real( -11.0 / 54.0 ), Type::real( 5.0 / 2.0 ), Type::real( -70.0 / 27.0 ), Type::real( 35.0 / 27.0 ) );

                         CALCULATE_K( 5, Type::real( 1.0 ), buffer_wavefunction, buffer_reservoir );

                         INTERMEDIATE_SUM_K( 5, Type::real( 1631.0 / 55296.0 ), Type::real( 175.0 / 512.0 ), Type::real( 575.0 / 13824.0 ), Type::real( 44275.0 / 110592.0 ), Type::real( 253.0 / 4096.0 ) );

                         CALCULATE_K( 6, Type::real( 7.0 / 8.0 ), buffer_wavefunction, buffer_reservoir );

                         INTERMEDIATE_SUM_K( 6, Type::real( 37.0 / 378.0 ), Type::real( 0.0 ), Type::real( 250.0 / 621.0 ), Type::real( 125.0 / 594.0 ), Type::real( 0.0 ), Type::real( 512.0 / 1771.0 ) );

                         ERROR_K( 6, Type::real( 37.0 / 378.0 - 2825.0 / 27648.0 ), Type::real( 0.0 ), Type::real( 250.0 / 621.0 - 18575.0 / 48384.0 ), Type::real( 125.0 / 594.0 - 13525.0 / 55296.0 ), Type::real( 0.0 - 2187.0 / 6784.0 ), Type::real( 512.0 / 1771.0 - 1.0 / 4.0 ) );

        );

        auto msum = matrix.buffer_wavefunction_plus.transformReduce( Type::complex( 0.0 ), CUDAMatrix<Type::complex>::transform_abs2(), CUDAMatrix<Type::complex>::transform_sum() );
        Type::real normalization_factor = CUDA::real( msum );
        Type::real integrated_error = matrix.rk_error.sum();

        Type::real final_error = std::abs( integrated_error / normalization_factor );
        Type::real dh_arg = system.tolerance / 2.0 / CUDA::max( std::numeric_limits<Type::real>::min(), final_error );
        Type::real dh = std::pow<Type::real>( dh_arg, Type::real( 0.25 ) );

        if ( std::isnan( dh ) ) {
            dh = 0.9;
            final_error = std::numeric_limits<Type::real>::max();
        }
        if ( std::isnan( final_error ) ) {
            dh = 0.9;
            final_error = std::numeric_limits<Type::real>::max();
        }

        //  Set new timestep
        Type::real new_dt = std::min( system.p.dt * dh, system.dt_max );
        if ( new_dt < system.dt_min ) {
            new_dt = system.dt_min;
            accept = true;
        }
        system.p.dt = new_dt;

        updateKernelTime();

        if ( final_error < system.tolerance ) {
            accept = true;
            matrix.wavefunction_plus.swap( matrix.buffer_wavefunction_plus );
            matrix.reservoir_plus.swap( matrix.buffer_reservoir_plus );
            if ( system.use_twin_mode ) {
                matrix.wavefunction_minus.swap( matrix.buffer_wavefunction_minus );
                matrix.reservoir_minus.swap( matrix.buffer_reservoir_minus );
            }
        }

    } while ( !accept );
}