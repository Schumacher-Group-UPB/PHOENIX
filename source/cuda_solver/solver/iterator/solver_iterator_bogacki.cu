#include "hip/hip_runtime.h"
#include <omp.h>

// Include Cuda Kernel headers
#include "cuda/typedef.cuh"
#include "kernel/kernel_compute.cuh"
#include "kernel/kernel_summation.cuh"
#include "kernel/kernel_halo.cuh"
#include "system/system_parameters.hpp"
#include "cuda/cuda_matrix.cuh"
#include "solver/gpu_solver.hpp"
#include "misc/commandline_io.hpp"

void PHOENIX::Solver::iterateFixedTimestepBogacki() {
    SOLVER_SEQUENCE( true /*Capture CUDA Graph*/,

                     CALCULATE_K( 1, Type::real( 0.0 ), wavefunction, reservoir );

                     INTERMEDIATE_SUM_K( 1, Type::real( 1.0 / 2.0 ) );

                     CALCULATE_K( 2, Type::real( 1.0 / 2.0 ), buffer_wavefunction, buffer_reservoir );

                     INTERMEDIATE_SUM_K( 2, Type::real( 0.0 ), Type::real( 3.0 / 9.0 ) );

                     CALCULATE_K( 3, Type::real( 3.0 / 4.0 ), buffer_wavefunction, buffer_reservoir );

                     FINAL_SUM_K( 3, Type::real( 2.0 / 9.0 ), Type::real( 1.0 / 3.0 ), Type::real( 4.0 / 9.0 ) );

    );
}

void PHOENIX::Solver::iterateVariableTimestepBogacki() {
    bool accept = false;
    do {
        SOLVER_SEQUENCE( false /*Capture CUDA Graph*/,

                         CALCULATE_K( 1, Type::real( 0.0 ), wavefunction, reservoir );

                         INTERMEDIATE_SUM_K( 1, Type::real( 1.0 / 2.0 ) );

                         CALCULATE_K( 2, Type::real( 1.0 / 2.0 ), buffer_wavefunction, buffer_reservoir );

                         INTERMEDIATE_SUM_K( 2, Type::real( 0.0 ), Type::real( 3.0 / 9.0 ) );

                         CALCULATE_K( 3, Type::real( 3.0 / 4.0 ), buffer_wavefunction, buffer_reservoir );

                         INTERMEDIATE_SUM_K( 3, Type::real( 2.0 / 9.0 ), Type::real( 1.0 / 3.0 ), Type::real( 4.0 / 9.0 ) );

                         // We need the 4th K for the error estimate
                         CALCULATE_K( 4, Type::real( 1.0 ), buffer_wavefunction, buffer_reservoir );

                         ERROR_K( 4, Type::real( 2.0 / 9.0 - 7.0 / 24.0 ), Type::real( 1.0 / 3.0 - 1.0 / 4.0 ), Type::real( 4.0 / 9.0 - 1.0 / 3.0 ), Type::real( 0.0 - 1.0 / 8.0 ) );

                         // Redo this sum so we get the correct solution in buffer_...
                         INTERMEDIATE_SUM_K( 3, Type::real( 2.0 / 9.0 ), Type::real( 1.0 / 3.0 ), Type::real( 4.0 / 9.0 ) ); );

        auto msum = matrix.buffer_wavefunction_plus.transformReduce( Type::complex( 0.0 ), CUDAMatrix<Type::complex>::transform_abs2(), CUDAMatrix<Type::complex>::transform_sum() );
        Type::real normalization_factor = CUDA::real( msum );
        Type::real integrated_error = matrix.rk_error.sum();

        Type::real final_error = std::abs( integrated_error / normalization_factor );
        Type::real dh_arg = system.tolerance / 2.0 / CUDA::max( std::numeric_limits<Type::real>::min(), final_error );
        Type::real dh = std::pow<Type::real>( dh_arg, Type::real( 1.0 / 4.0 ) );

        if ( std::isnan( dh ) ) {
            dh = 0.9;
            final_error = std::numeric_limits<Type::real>::max();
        }
        if ( std::isnan( final_error ) ) {
            dh = 0.9;
            final_error = std::numeric_limits<Type::real>::max();
        }

        //  Set new timestep
        Type::real new_dt = std::min( system.p.dt * dh, system.dt_max );
        if ( new_dt < system.dt_min ) {
            new_dt = system.dt_min;
            accept = true;
        }
        system.p.dt = new_dt;

        updateKernelTime();

        if ( final_error < system.tolerance ) {
            accept = true;
            matrix.wavefunction_plus.swap( matrix.buffer_wavefunction_plus );
            matrix.reservoir_plus.swap( matrix.buffer_reservoir_plus );
            if ( system.use_twin_mode ) {
                matrix.wavefunction_minus.swap( matrix.buffer_wavefunction_minus );
                matrix.reservoir_minus.swap( matrix.buffer_reservoir_minus );
            }
        }
    } while ( !accept );
}