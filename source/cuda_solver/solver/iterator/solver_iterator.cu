#include "hip/hip_runtime.h"
#include <omp.h>

// Include Cuda Kernel headers
#include "cuda/typedef.cuh"
#include "kernel/kernel_compute.cuh"
#include "system/system_parameters.hpp"
#include "cuda/cuda_matrix.cuh"
#include "solver/gpu_solver.hpp"
#include "misc/commandline_io.hpp"

/*
* Helper variable for caching the current time for FFT evaluations.
* We dont need this variable anywhere else, so we just create it
* locally to this file here.
*/
PHOENIX::Type::real fft_cached_t = 0.0;
bool first_time = true;

void PHOENIX::Solver::updateKernelTime() {
    // Update the time struct. This is required for variable time steps, and when the kernels need t or dt.
    Type::host_vector<Type::real> new_time = { system.p.t, system.p.dt };
    time = new_time;
    // And update the solver struct accordingly
    system.pulse.updateTemporal( system.p.t );
    system.potential.updateTemporal( system.p.t );
    system.pump.updateTemporal( system.p.t );
    dev_pulse_oscillation.amp = system.pulse.temporal_envelope;
    dev_potential_oscillation.amp = system.potential.temporal_envelope;
    dev_pump_oscillation.amp = system.pump.temporal_envelope;
    system.pulse.updateTemporal( system.p.t + system.p.dt );
    system.potential.updateTemporal( system.p.t + system.p.dt );
    system.pump.updateTemporal( system.p.t + system.p.dt );
    dev_pulse_oscillation.amp_next = system.pulse.temporal_envelope;
    dev_potential_oscillation.amp_next = system.potential.temporal_envelope;
    dev_pump_oscillation.amp_next = system.pump.temporal_envelope;
}

bool PHOENIX::Solver::iterate() {
    // Check if the maximum time has been reached
#ifndef BENCH
    if ( system.p.t >= system.t_max )
        return false;
#endif

    // If required, calculate new set of random numbers.
    // TODO: move this back into subgrids, because for large number of subgrids this will look very correlated!
    if ( system.evaluateStochastic() ) {
        auto args = generateKernelArguments();
        auto [block_size, grid_size] = getLaunchParameters( 1, system.p.subgrid_N2_with_halo );
        if ( first_time ) {
            first_time = false;
            CALL_FULL_KERNEL( PHOENIX::Kernel::initialize_random_number_generator, "random_number_init", grid_size, block_size, 0, system.random_seed, args.dev_ptrs.random_state, system.p.subgrid_N2_with_halo );
            std::cout << PHOENIX::CLIO::prettyPrint( "Initialized Random Number Generator", PHOENIX::CLIO::Control::Info ) << std::endl;
        }
        CALL_FULL_KERNEL( PHOENIX::Kernel::generate_random_numbers, "random_number_gen", grid_size, block_size, 0, args.dev_ptrs.random_state, args.dev_ptrs.random_number, system.p.subgrid_N2_with_halo, system.p.stochastic_amplitude * std::sqrt( system.p.dt ), system.p.stochastic_amplitude * std::sqrt( system.p.dt ) );
    }
    
    updateKernelTime();

    // Increase t.
    system.p.t = system.p.t + system.p.dt;

    // Iterate RK4(45)/ssfm/itp
    iterator[system.iterator].iterate();

    // Call the normalization for imaginary time propagation if required
    if ( system.imag_time_amplitude != 0.0 )
        normalizeImaginaryTimePropagation();

    // For statistical purposes, increase the iteration counter
    system.iteration++;

    // FFT Guard
    if ( system.p.t - fft_cached_t < system.fft_every )
        return true;

    // Calculate the FFT
    fft_cached_t = system.p.t;
    applyFFTFilter( system.fft_mask.size() > 0 );

    return true;
}
