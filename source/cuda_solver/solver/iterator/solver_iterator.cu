#include "hip/hip_runtime.h"
#include <omp.h>

// Include Cuda Kernel headers
#include "cuda/typedef.cuh"
#include "kernel/kernel_compute.cuh"
#include "system/system_parameters.hpp"
#include "cuda/cuda_matrix.cuh"
#include "solver/gpu_solver.hpp"
#include "misc/commandline_io.hpp"

/*
* Helper variable for caching the current time for FFT evaluations.
* We dont need this variable anywhere else, so we just create it
* locally to this file here.
*/
PC3::Type::real fft_cached_t = 0.0;
bool first_time = true;

/**
 * Iterates the Runge-Kutta-Method on the GPU
 * Note, that all device arrays and variables have to be initialized at this point
 * @param t Current time, will be updated to t + dt
 * @param dt Time step, will be updated to the next time step
 * @param N_c Number of grid points in one dimension
 * @param N_r Number of grid points in the other dimension
 */
bool PC3::Solver::iterate() {
    // First, check if the maximum time has been reached
    if ( system.p.t >= system.t_max )
        return false;

    // If required, calculate new set of random numbers.
    // TODO: move this back into subgrids, because for large number of subgrids this will look very correlated!
    if (system.evaluateStochastic()) {
        auto args = generateKernelArguments( );
        auto [block_size, grid_size] = getLaunchParameters( system.p.subgrid_N_c, system.p.subgrid_N_r );
        if (first_time) {
            first_time = false;
            CALL_FULL_KERNEL(
                    PC3::Kernel::initialize_random_number_generator, "random_number_init", grid_size, block_size, 0,
                    system.random_seed, args.dev_ptrs.random_state, system.p.subgrid_N_c*system.p.subgrid_N_r
                );
            std::cout << PC3::CLIO::prettyPrint( "Initialized Random Number Generator", PC3::CLIO::Control::Info ) << std::endl;
        }
        CALL_FULL_KERNEL(
            PC3::Kernel::generate_random_numbers, "random_number_gen", grid_size, block_size, 0,
            args.dev_ptrs.random_state, args.dev_ptrs.random_number, system.p.subgrid_N_c*system.p.subgrid_N_r, system.p.stochastic_amplitude*std::sqrt(system.p.dt), system.p.stochastic_amplitude*std::sqrt(system.p.dt)
        );
    }

    // TODO: Merhe these device arrays with the kernelParameters struct.
    // should be easily possible because the sizes of the arrays are known at launch
    // which means we can allocate the memory in the kernelParameters struct
    // Update the temporal envelopes
    // measure time taken
    system.pulse.updateTemporal( system.p.t );
    system.potential.updateTemporal( system.p.t );
    system.pump.updateTemporal( system.p.t );
    // And update the solver struct accordingly
    dev_pulse_oscillation.amp = system.pulse.temporal_envelope;
    dev_potential_oscillation.amp = system.potential.temporal_envelope;
    dev_pump_oscillation.amp = system.pump.temporal_envelope;

    // Iterate RK4(45)/ssfm/itp
    iterator[system.iterator].iterate();

    // Call the normalization for imaginary time propagation if required
    if ( system.imag_time_amplitude != 0.0 )
        normalizeImaginaryTimePropagation();

    // Increase t.
    system.p.t = system.p.t + system.p.dt;

    // For statistical purposes, increase the iteration counter
    system.iteration++;

    // FFT Guard
    if ( system.p.t - fft_cached_t < system.fft_every )
        return true;

    // Calculate the FFT
    fft_cached_t = system.p.t;
    applyFFTFilter( system.fft_mask.size() > 0 );

    return true;
}