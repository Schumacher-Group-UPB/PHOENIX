#include "hip/hip_runtime.h"
#include <omp.h>

// Include Cuda Kernel headers
#include "cuda/typedef.cuh"
#include "kernel/kernel_compute.cuh"
#include "kernel/kernel_summation.cuh"
#include "kernel/kernel_halo.cuh"
#include "system/system_parameters.hpp"
#include "cuda/cuda_matrix.cuh"
#include "solver/gpu_solver.hpp"
#include "misc/commandline_io.hpp"

void PHOENIX::Solver::iterateFixedTimestepDOP5() {
    SOLVER_SEQUENCE( true /*Capture CUDA Graph*/,

                     CALCULATE_K( 1, Type::real( 0.0 ), wavefunction, reservoir );

                     INTERMEDIATE_SUM_K( 1, Type::real( 1.0 / 5.0 ) );

                     CALCULATE_K( 2, Type::real( 1.0 / 5.0 ), buffer_wavefunction, buffer_reservoir );

                     INTERMEDIATE_SUM_K( 2, Type::real( 3.0 / 40.0 ), Type::real( 9.0 / 40.0 ) );

                     CALCULATE_K( 3, Type::real( 3.0 / 10.0 ), buffer_wavefunction, buffer_reservoir );

                     INTERMEDIATE_SUM_K( 3, Type::real( 44.0 / 45.0 ), Type::real( -56.0 / 15.0 ), Type::real( 32.0 / 9.0 ) );

                     CALCULATE_K( 4, Type::real( 4.0 / 5.0 ), buffer_wavefunction, buffer_reservoir );

                     INTERMEDIATE_SUM_K( 4, Type::real( 19372.0 / 6561.0 ), Type::real( -25360.0 / 2187.0 ), Type::real( 64448.0 / 6561.0 ), Type::real( -212.0 / 729.0 ) );

                     CALCULATE_K( 5, Type::real( 8.0 / 9.0 ), buffer_wavefunction, buffer_reservoir );

                     INTERMEDIATE_SUM_K( 5, Type::real( 9017.0 / 3168.0 ), Type::real( -355.0 / 33.0 ), Type::real( 46732.0 / 5247.0 ), Type::real( 49.0 / 176.0 ), Type::real( -5103.0 / 18656.0 ) );

                     CALCULATE_K( 6, Type::real( 1.0 ), buffer_wavefunction, buffer_reservoir );

                     FINAL_SUM_K( 6, Type::real( 35.0 / 384.0 ), Type::real( 0.0 ), Type::real( 500.0 / 1113.0 ), Type::real( 125.0 / 192.0 ), Type::real( -2187.0 / 6784.0 ), Type::real( 11.0 / 84.0 ) );

    );
}

void PHOENIX::Solver::iterateVariableTimestepDOP45() {
    bool accept = false;
    do {
        SOLVER_SEQUENCE( false /*Capture CUDA Graph*/,

                         CALCULATE_K( 1, Type::real( 0.0 ), wavefunction, reservoir );

                         INTERMEDIATE_SUM_K( 1, Type::real( 1.0 / 5.0 ) );

                         CALCULATE_K( 2, Type::real( 1.0 / 5.0 ), buffer_wavefunction, buffer_reservoir );

                         INTERMEDIATE_SUM_K( 2, Type::real( 3.0 / 40.0 ), Type::real( 9.0 / 40.0 ) );

                         CALCULATE_K( 3, Type::real( 3.0 / 10.0 ), buffer_wavefunction, buffer_reservoir );

                         INTERMEDIATE_SUM_K( 3, Type::real( 44.0 / 45.0 ), Type::real( -56.0 / 15.0 ), Type::real( 32.0 / 9.0 ) );

                         CALCULATE_K( 4, Type::real( 4.0 / 5.0 ), buffer_wavefunction, buffer_reservoir );

                         INTERMEDIATE_SUM_K( 4, Type::real( 19372.0 / 6561.0 ), Type::real( -25360.0 / 2187.0 ), Type::real( 64448.0 / 6561.0 ), Type::real( -212.0 / 729.0 ) );

                         CALCULATE_K( 5, Type::real( 8.0 / 9.0 ), buffer_wavefunction, buffer_reservoir );

                         INTERMEDIATE_SUM_K( 5, Type::real( 9017.0 / 3168.0 ), Type::real( -355.0 / 33.0 ), Type::real( 46732.0 / 5247.0 ), Type::real( 49.0 / 176.0 ), Type::real( -5103.0 / 18656.0 ) );

                         CALCULATE_K( 6, Type::real( 1.0 ), buffer_wavefunction, buffer_reservoir );

                         INTERMEDIATE_SUM_K( 6, Type::real( 35.0 / 384.0 ), Type::real( 0.0 ), Type::real( 500.0 / 1113.0 ), Type::real( 125.0 / 192.0 ), Type::real( -2187.0 / 6784.0 ), Type::real( 11.0 / 84.0 ) );

                         // For DP, we need the 7th k
                         CALCULATE_K( 7, Type::real( 1.0 ), buffer_wavefunction, buffer_reservoir );

                         ERROR_K( 7, Type::real( 35.0 / 384.0 - 5179.0 / 57600.0 ), Type::real( 0.0 ), Type::real( 500.0 / 1113.0 - 7571.0 / 16695.0 ), Type::real( 125.0 / 192.0 - 393.0 / 640.0 ), Type::real( -2187.0 / 6784.0 + 92097.0 / 339200.0 ), Type::real( 11.0 / 84.0 - 187.0 / 2100.0 ), Type::real( -1.0 / 40.0 ) );

                         // Redo this sum so we get the correct solution in buffer_...
                         INTERMEDIATE_SUM_K( 6, Type::real( 35.0 / 384.0 ), Type::real( 0.0 ), Type::real( 500.0 / 1113.0 ), Type::real( 125.0 / 192.0 ), Type::real( -2187.0 / 6784.0 ), Type::real( 11.0 / 84.0 ) );

        );
        auto msum = matrix.k_wavefunction_plus.transformReduce( Type::complex( 0.0 ), CUDAMatrix<Type::complex>::transform_abs2(), CUDAMatrix<Type::complex>::transform_sum(), 5 /*matrix k6*/ );
        Type::real normalization_factor = CUDA::real( msum );
        Type::real integrated_error = matrix.rk_error.sum();

        Type::real final_error = std::abs( integrated_error / normalization_factor );
        Type::real dh_arg = system.tolerance / 2.0 / CUDA::max( std::numeric_limits<Type::real>::min(), final_error );
        Type::real dh = std::pow<Type::real>( dh_arg, Type::real( 0.25 ) );

        if ( std::isnan( dh ) ) {
            dh = 0.9;
            final_error = std::numeric_limits<Type::real>::max();
        }
        if ( std::isnan( final_error ) ) {
            dh = 0.9;
            final_error = std::numeric_limits<Type::real>::max();
        }

        //  Set new timestep
        Type::real new_dt = std::min( system.p.dt * dh, system.dt_max );
        if ( new_dt < system.dt_min ) {
            new_dt = system.dt_min;
            accept = true;
        }
        system.p.dt = new_dt;

        updateKernelTime();

        if ( final_error < system.tolerance ) {
            accept = true;
            matrix.wavefunction_plus.swap( matrix.buffer_wavefunction_plus );
            matrix.reservoir_plus.swap( matrix.buffer_reservoir_plus );
            if ( system.use_twin_mode ) {
                matrix.wavefunction_minus.swap( matrix.buffer_wavefunction_minus );
                matrix.reservoir_minus.swap( matrix.buffer_reservoir_minus );
            }
        }

    } while ( !accept );
}

void PHOENIX::Solver::iterateFixedTimestepDOP853() {
    SOLVER_SEQUENCE( true /*Capture CUDA Graph*/,

                     // Stage 1 (t + 0)
                     CALCULATE_K( 1, Type::real( 0.0 ), wavefunction, reservoir );

                     INTERMEDIATE_SUM_K( 1, Type::real( 5.260015195876773e-2 ) );

                     CALCULATE_K( 2, Type::real( 0.05260015196 ), buffer_wavefunction, buffer_reservoir );

                     INTERMEDIATE_SUM_K( 2, Type::real( 1.9725056984537899e-2 ), Type::real( 5.9175170953613698e-2 ) );

                     CALCULATE_K( 3, Type::real( 0.07890022794 ), buffer_wavefunction, buffer_reservoir );

                     INTERMEDIATE_SUM_K( 3, Type::real( 2.9587585476806849e-2 ), Type::real( 0.0 ), Type::real( 8.8762756430420548e-2 ) );

                     CALCULATE_K( 4, Type::real( 0.1183503419 ), buffer_wavefunction, buffer_reservoir );

                     INTERMEDIATE_SUM_K( 4, Type::real( 2.4136513415926669e-1 ), Type::real( 0.0 ), Type::real( -8.8454947932828610e-1 ), Type::real( 9.2483400326179200e-1 ) );

                     CALCULATE_K( 5, Type::real( 0.2816496581 ), buffer_wavefunction, buffer_reservoir );

                     INTERMEDIATE_SUM_K( 5, Type::real( 3.7037037037037037e-2 ), Type::real( 0.0 ), Type::real( 0.0 ), Type::real( 1.7082860872947387e-1 ), Type::real( 1.2546768756682243e-1 ) );

                     CALCULATE_K( 6, Type::real( 0.3333333333 ), buffer_wavefunction, buffer_reservoir );

                     INTERMEDIATE_SUM_K( 6, Type::real( 3.7109375e-2 ), Type::real( 0.0 ), Type::real( 0.0 ), Type::real( 1.7025221101954404e-1 ), Type::real( 6.0216538980455961e-2 ), Type::real( -1.7578125e-2 ) );

                     CALCULATE_K( 7, Type::real( 0.25 ), buffer_wavefunction, buffer_reservoir );

                     INTERMEDIATE_SUM_K( 7, Type::real( 3.7092000118504793e-2 ), Type::real( 0.0 ), Type::real( 0.0 ), Type::real( 1.7038392571223999e-1 ), Type::real( 1.0726203044637328e-1 ), Type::real( -1.5319437748624402e-2 ), Type::real( 8.2737891638140229e-3 ) );

                     CALCULATE_K( 8, Type::real( 0.3076923077 ), buffer_wavefunction, buffer_reservoir );

                     INTERMEDIATE_SUM_K( 8, Type::real( 6.2411095871607572e-1 ), Type::real( 0.0 ), Type::real( 0.0 ), Type::real( -3.3608926294469413e+0 ), Type::real( -8.6821934684172601e-1 ), Type::real( 2.7592099699446708e+1 ), Type::real( 2.0154067550477893e+1 ), Type::real( -4.3489884181069959e+1 ) );

                     CALCULATE_K( 9, Type::real( 0.6512820513 ), buffer_wavefunction, buffer_reservoir );

                     INTERMEDIATE_SUM_K( 9, Type::real( 4.7766253643826437e-1 ), Type::real( 0.0 ), Type::real( 0.0 ), Type::real( -2.4881146199716676e+0 ), Type::real( -5.9029082683684300e-1 ), Type::real( 2.1230051448181194e+1 ), Type::real( 1.5279233632882424e+1 ), Type::real( -3.3288210968984863e+1 ), Type::real( -2.0331201708508626e-2 ) );

                     CALCULATE_K( 10, Type::real( 0.6 ), buffer_wavefunction, buffer_reservoir );

                     INTERMEDIATE_SUM_K( 10, Type::real( -9.3714243008598733e-1 ), Type::real( 0.0 ), Type::real( 0.0 ), Type::real( 5.1863724288440637e+0 ), Type::real( 1.0914373489967296e+0 ), Type::real( -8.1497870107469261e+0 ), Type::real( -1.8520065659996960e+1 ), Type::real( 2.2739487099350504e+1 ), Type::real( 2.4936055526796524e+0 ), Type::real( -3.0467644718982195e+0 ) );

                     CALCULATE_K( 11, Type::real( 0.8571428571 ), buffer_wavefunction, buffer_reservoir );

                     INTERMEDIATE_SUM_K( 11, Type::real( 2.2733101475165382e+0 ), Type::real( 0.0 ), Type::real( 0.0 ), Type::real( -1.0534495466737250e+1 ), Type::real( -2.0008720582248625e+0 ), Type::real( -1.7958931863118799e+1 ), Type::real( 2.7948884529419960e+1 ), Type::real( -2.8589982771350237e+0 ), Type::real( -8.8728569335306295e+0 ), Type::real( 1.2360567175794303e+1 ), Type::real( 6.4339274601576353e-1 ) );

                     CALCULATE_K( 12, Type::real( 1.0 ), buffer_wavefunction, buffer_reservoir );

                     FINAL_SUM_K( 12, Type::real( 5.4293734116568762e-2 ), Type::real( 0.0 ), Type::real( 0.0 ), Type::real( 0.0 ), Type::real( 0.0 ), Type::real( 4.4503128927524089e+0 ), Type::real( 1.8915178993145004e+0 ), Type::real( -5.8012039600105848e+0 ), Type::real( 3.1116436695781989e-1 ), Type::real( -1.5216094966251608e-1 ), Type::real( 2.0136540080403035e-1 ), Type::real( 4.4710615727772591e-2 ) );

    );
}

void PHOENIX::Solver::iterateVariableTimestepDOP853() {
    bool accept = false;
    do {
        SOLVER_SEQUENCE( false /*Capture CUDA Graph*/,

                         CALCULATE_K( 1, Type::real( 0.0 ), wavefunction, reservoir );

                         INTERMEDIATE_SUM_K( 1, Type::real( 5.260015195876773e-2 ) );

                         CALCULATE_K( 2, Type::real( 0.05260015196 ), buffer_wavefunction, buffer_reservoir );

                         INTERMEDIATE_SUM_K( 2, Type::real( 1.9725056984537899e-2 ), Type::real( 5.9175170953613698e-2 ) );

                         CALCULATE_K( 3, Type::real( 0.07890022794 ), buffer_wavefunction, buffer_reservoir );

                         INTERMEDIATE_SUM_K( 3, Type::real( 2.9587585476806849e-2 ), Type::real( 0.0 ), Type::real( 8.8762756430420548e-2 ) );

                         CALCULATE_K( 4, Type::real( 0.1183503419 ), buffer_wavefunction, buffer_reservoir );

                         INTERMEDIATE_SUM_K( 4, Type::real( 2.4136513415926669e-1 ), Type::real( 0.0 ), Type::real( -8.8454947932828610e-1 ), Type::real( 9.2483400326179200e-1 ) );

                         CALCULATE_K( 5, Type::real( 0.2816496581 ), buffer_wavefunction, buffer_reservoir );

                         INTERMEDIATE_SUM_K( 5, Type::real( 3.7037037037037037e-2 ), Type::real( 0.0 ), Type::real( 0.0 ), Type::real( 1.7082860872947387e-1 ), Type::real( 1.2546768756682243e-1 ) );

                         CALCULATE_K( 6, Type::real( 0.3333333333 ), buffer_wavefunction, buffer_reservoir );

                         INTERMEDIATE_SUM_K( 6, Type::real( 3.7109375e-2 ), Type::real( 0.0 ), Type::real( 0.0 ), Type::real( 1.7025221101954404e-1 ), Type::real( 6.0216538980455961e-2 ), Type::real( -1.7578125e-2 ) );

                         CALCULATE_K( 7, Type::real( 0.25 ), buffer_wavefunction, buffer_reservoir );

                         INTERMEDIATE_SUM_K( 7, Type::real( 3.7092000118504793e-2 ), Type::real( 0.0 ), Type::real( 0.0 ), Type::real( 1.7038392571223999e-1 ), Type::real( 1.0726203044637328e-1 ), Type::real( -1.5319437748624402e-2 ), Type::real( 8.2737891638140229e-3 ) );

                         CALCULATE_K( 8, Type::real( 0.3076923077 ), buffer_wavefunction, buffer_reservoir );

                         INTERMEDIATE_SUM_K( 8, Type::real( 6.2411095871607572e-1 ), Type::real( 0.0 ), Type::real( 0.0 ), Type::real( -3.3608926294469413e+0 ), Type::real( -8.6821934684172601e-1 ), Type::real( 2.7592099699446708e+1 ), Type::real( 2.0154067550477893e+1 ), Type::real( -4.3489884181069959e+1 ) );

                         CALCULATE_K( 9, Type::real( 0.6512820513 ), buffer_wavefunction, buffer_reservoir );

                         INTERMEDIATE_SUM_K( 9, Type::real( 4.7766253643826437e-1 ), Type::real( 0.0 ), Type::real( 0.0 ), Type::real( -2.4881146199716676e+0 ), Type::real( -5.9029082683684300e-1 ), Type::real( 2.1230051448181194e+1 ), Type::real( 1.5279233632882424e+1 ), Type::real( -3.3288210968984863e+1 ), Type::real( -2.0331201708508626e-2 ) );

                         CALCULATE_K( 10, Type::real( 0.6 ), buffer_wavefunction, buffer_reservoir );

                         INTERMEDIATE_SUM_K( 10, Type::real( -9.3714243008598733e-1 ), Type::real( 0.0 ), Type::real( 0.0 ), Type::real( 5.1863724288440637e+0 ), Type::real( 1.0914373489967296e+0 ), Type::real( -8.1497870107469261e+0 ), Type::real( -1.8520065659996960e+1 ), Type::real( 2.2739487099350504e+1 ), Type::real( 2.4936055526796524e+0 ), Type::real( -3.0467644718982195e+0 ) );

                         CALCULATE_K( 11, Type::real( 0.8571428571 ), buffer_wavefunction, buffer_reservoir );

                         INTERMEDIATE_SUM_K( 11, Type::real( 2.2733101475165382e+0 ), Type::real( 0.0 ), Type::real( 0.0 ), Type::real( -1.0534495466737250e+1 ), Type::real( -2.0008720582248625e+0 ), Type::real( -1.7958931863118799e+1 ), Type::real( 2.7948884529419960e+1 ), Type::real( -2.8589982771350237e+0 ), Type::real( -8.8728569335306295e+0 ), Type::real( 1.2360567175794303e+1 ), Type::real( 6.4339274601576353e-1 ) );

                         CALCULATE_K( 12, Type::real( 1.0 ), buffer_wavefunction, buffer_reservoir );

                         INTERMEDIATE_SUM_K( 12, Type::real( 5.4293734116568762e-2 ), Type::real( 0.0 ), Type::real( 0.0 ), Type::real( 0.0 ), Type::real( 0.0 ), Type::real( 4.4503128927524089e+0 ), Type::real( 1.8915178993145004e+0 ), Type::real( -5.8012039600105848e+0 ), Type::real( 3.1116436695781989e-1 ), Type::real( -1.5216094966251608e-1 ), Type::real( 2.0136540080403035e-1 ), Type::real( 4.4710615727772591e-2 ) );

                         ERROR_K( 12, Type::real( 4.1173689122373888e-2 ), Type::real( 0.0 ), Type::real( 0.0 ), Type::real( 0.0 ), Type::real( 0.0 ), Type::real( 5.6754693391286133e+0 ), Type::real( 2.3872768489717506e+0 ), Type::real( -7.4655811424655713e+0 ), Type::real( 6.6149321570779360e-1 ), Type::real( -4.8634006837553356e-1 ), Type::real( 1.1944219431891464e-1 ), Type::real( 6.7065923591658886e-2 ) );

        );
        auto msum = matrix.buffer_wavefunction_plus.transformReduce( Type::complex( 0.0 ), CUDAMatrix<Type::complex>::transform_abs2(), CUDAMatrix<Type::complex>::transform_sum() );
        Type::real normalization_factor = CUDA::real( msum );
        Type::real integrated_error = matrix.rk_error.sum();

        Type::real final_error = std::abs( integrated_error / normalization_factor );
        Type::real dh_arg = system.tolerance / 2.0 / CUDA::max( std::numeric_limits<Type::real>::min(), final_error );
        Type::real dh = std::pow<Type::real>( dh_arg, Type::real( 0.16 ) );

        if ( std::isnan( dh ) ) {
            dh = 0.9;
            final_error = std::numeric_limits<Type::real>::max();
        }
        if ( std::isnan( final_error ) ) {
            dh = 0.9;
            final_error = std::numeric_limits<Type::real>::max();
        }

        //  Set new timestep
        Type::real new_dt = std::min( system.p.dt * dh, system.dt_max );
        if ( new_dt < system.dt_min ) {
            new_dt = system.dt_min;
            accept = true;
        }
        system.p.dt = new_dt;

        updateKernelTime();

        if ( final_error < system.tolerance ) {
            accept = true;
            matrix.wavefunction_plus.swap( matrix.buffer_wavefunction_plus );
            matrix.reservoir_plus.swap( matrix.buffer_reservoir_plus );
            if ( system.use_twin_mode ) {
                matrix.wavefunction_minus.swap( matrix.buffer_wavefunction_minus );
                matrix.reservoir_minus.swap( matrix.buffer_reservoir_minus );
            }
        }
    } while ( !accept );
}