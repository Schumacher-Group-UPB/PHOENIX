#include "hip/hip_runtime.h"
#include <omp.h>

// Include Cuda Kernel headers
#include "cuda/typedef.cuh"
#include "kernel/kernel_compute.cuh"
#include "kernel/kernel_summation.cuh"
#include "kernel/kernel_halo.cuh"
#include "system/system_parameters.hpp"
#include "cuda/cuda_matrix.cuh"
#include "solver/gpu_solver.hpp"
#include "misc/commandline_io.hpp"

void PHOENIX::Solver::iterateFixedTimestepDOP5() {
    SOLVER_SEQUENCE( true /*Capture CUDA Graph*/,

                     CALCULATE_K( 1, wavefunction, reservoir );

                     INTERMEDIATE_SUM_K( 1, Type::real( 1.0 / 5.0 ) );

                     CALCULATE_K( 2, buffer_wavefunction, buffer_reservoir );

                     INTERMEDIATE_SUM_K( 2, Type::real( 3.0 / 40.0 ), Type::real( 9.0 / 40.0 ) );

                     CALCULATE_K( 3, buffer_wavefunction, buffer_reservoir );

                     INTERMEDIATE_SUM_K( 3, Type::real( 44.0 / 45.0 ), Type::real( -56.0 / 15.0 ), Type::real( 32.0 / 9.0 ) );

                     CALCULATE_K( 4, buffer_wavefunction, buffer_reservoir );

                     INTERMEDIATE_SUM_K( 4, Type::real( 19372.0 / 6561.0 ), Type::real( -25360.0 / 2187.0 ), Type::real( 64448.0 / 6561.0 ), Type::real( -212.0 / 729.0 ) );

                     CALCULATE_K( 5, buffer_wavefunction, buffer_reservoir );

                     INTERMEDIATE_SUM_K( 5, Type::real( 9017.0 / 3168.0 ), Type::real( -355.0 / 33.0 ), Type::real( 46732.0 / 5247.0 ), Type::real( 49.0 / 176.0 ), Type::real( -5103.0 / 18656.0 ) );

                     CALCULATE_K( 6, buffer_wavefunction, buffer_reservoir );

                     FINAL_SUM_K( 6, Type::real( 35.0 / 384.0 ), Type::real( 0.0 ), Type::real( 500.0 / 1113.0 ), Type::real( 125.0 / 192.0 ), Type::real( -2187.0 / 6784.0 ), Type::real( 11.0 / 84.0 ) );

    );
}

void PHOENIX::Solver::iterateVariableTimestepDOP45() {
    bool accept = false;
    do {
        SOLVER_SEQUENCE( false /*Capture CUDA Graph*/,

                         CALCULATE_K( 1, wavefunction, reservoir );

                         INTERMEDIATE_SUM_K( 1, Type::real( 1.0 / 5.0 ) );

                         CALCULATE_K( 2, buffer_wavefunction, buffer_reservoir );

                         INTERMEDIATE_SUM_K( 2, Type::real( 3.0 / 40.0 ), Type::real( 9.0 / 40.0 ) );

                         CALCULATE_K( 3, buffer_wavefunction, buffer_reservoir );

                         INTERMEDIATE_SUM_K( 3, Type::real( 44.0 / 45.0 ), Type::real( -56.0 / 15.0 ), Type::real( 32.0 / 9.0 ) );

                         CALCULATE_K( 4, buffer_wavefunction, buffer_reservoir );

                         INTERMEDIATE_SUM_K( 4, Type::real( 19372.0 / 6561.0 ), Type::real( -25360.0 / 2187.0 ), Type::real( 64448.0 / 6561.0 ), Type::real( -212.0 / 729.0 ) );

                         CALCULATE_K( 5, buffer_wavefunction, buffer_reservoir );

                         INTERMEDIATE_SUM_K( 5, Type::real( 9017.0 / 3168.0 ), Type::real( -355.0 / 33.0 ), Type::real( 46732.0 / 5247.0 ), Type::real( 49.0 / 176.0 ), Type::real( -5103.0 / 18656.0 ) );

                         CALCULATE_K( 6, buffer_wavefunction, buffer_reservoir );

                         INTERMEDIATE_SUM_K( 6, Type::real( 35.0 / 384.0 ), Type::real( 0.0 ), Type::real( 500.0 / 1113.0 ), Type::real( 125.0 / 192.0 ), Type::real( -2187.0 / 6784.0 ), Type::real( 11.0 / 84.0 ) );

                         // For DP, we need the 7th k
                         CALCULATE_K( 7, buffer_wavefunction, buffer_reservoir );

                         ERROR_K( 7, Type::real( 35.0 / 384.0 - 5179.0 / 57600.0 ), Type::real( 0.0 ), Type::real( 500.0 / 1113.0 - 7571.0 / 16695.0 ), Type::real( 125.0 / 192.0 - 393.0 / 640.0 ), Type::real( -2187.0 / 6784.0 + 92097.0 / 339200.0 ), Type::real( 11.0 / 84.0 - 187.0 / 2100.0 ), Type::real( -1.0 / 40.0 ) );

                         // Redo this sum so we get the correct solution in buffer_...
                         INTERMEDIATE_SUM_K( 6, Type::real( 35.0 / 384.0 ), Type::real( 0.0 ), Type::real( 500.0 / 1113.0 ), Type::real( 125.0 / 192.0 ), Type::real( -2187.0 / 6784.0 ), Type::real( 11.0 / 84.0 ) );

        );
        auto msum = matrix.k_wavefunction_plus.transformReduce( Type::complex( 0.0 ), CUDAMatrix<Type::complex>::transform_abs2(), CUDAMatrix<Type::complex>::transform_sum(), 5 /*matrix k6*/ );
        Type::real normalization_factor = CUDA::sqrt( CUDA::real( msum ) );
        Type::real integrated_error = std::sqrt( matrix.rk_error.sum() );

        Type::real final_error = std::abs( integrated_error / normalization_factor );
        Type::real dh_arg = system.tolerance / 2.0 / CUDA::max( std::numeric_limits<Type::real>::min(), final_error );
        Type::real dh = std::pow<Type::real>( dh_arg, Type::real( 1.0 / 5.0 ) );

        if ( std::isnan( dh ) ) {
            dh = 0.9;
            final_error = std::numeric_limits<Type::real>::max();
        }
        if ( std::isnan( final_error ) ) {
            dh = 0.9;
            final_error = std::numeric_limits<Type::real>::max();
        }

        //  Set new timestep
        Type::real new_dt = std::min( system.p.dt * dh, system.dt_max );
        if ( new_dt < system.dt_min ) {
            new_dt = system.dt_min;
            accept = true;
        }
        system.p.dt = new_dt;

        updateKernelTime();

        if ( final_error < system.tolerance ) {
            accept = true;
            matrix.wavefunction_plus.swap( matrix.buffer_wavefunction_plus );
            matrix.reservoir_plus.swap( matrix.buffer_reservoir_plus );
            if ( system.use_twin_mode ) {
                matrix.wavefunction_minus.swap( matrix.buffer_wavefunction_minus );
                matrix.reservoir_minus.swap( matrix.buffer_reservoir_minus );
            }
        }

    } while ( !accept );
}

void PHOENIX::Solver::iterateFixedTimestepDOP853() {
    SOLVER_SEQUENCE( true /*Capture CUDA Graph*/,

                     // ——— Dormand–Prince 8(5,3) (w) ———
                     // Coefficients taken verbatim from your dop853.f snippet.

                     // Stage 1 (t + 0)
                     CALCULATE_K( 1, wavefunction, reservoir );

                     // Stage 2 (t + c₂·h),   c₂ =  5.260015195876773e-2
                     INTERMEDIATE_SUM_K( 1, Type::real( 5.260015195876773e-2 ) ); CALCULATE_K( 2, buffer_wavefunction, buffer_reservoir );

                     // Stage 3 (t + c₃·h),   c₃ =  7.8900227938151598e-2
                     INTERMEDIATE_SUM_K( 2,
                                         Type::real( 1.9725056984537899e-2 ), // a₃₁
                                         Type::real( 5.9175170953613698e-2 )  // a₃₂
                     );
                     CALCULATE_K( 3, buffer_wavefunction, buffer_reservoir );

                     // Stage 4 (t + c₄·h),   c₄ =  1.1835034190722739e-1
                     INTERMEDIATE_SUM_K( 3,
                                         Type::real( 2.9587585476806849e-2 ), // a₄₁
                                         Type::real( 0.0 ),                   // a₄₂
                                         Type::real( 8.8762756430420548e-2 )  // a₄₃
                     );
                     CALCULATE_K( 4, buffer_wavefunction, buffer_reservoir );

                     // Stage 5 (t + c₅·h),   c₅ =  2.8164965809277260e-1
                     INTERMEDIATE_SUM_K( 4,
                                         Type::real( 2.4136513415926669e-1 ),  // a₅₁
                                         Type::real( 0.0 ),                    // a₅₂
                                         Type::real( -8.8454947932828610e-1 ), // a₅₃
                                         Type::real( 9.2483400326179200e-1 )   // a₅₄
                     );
                     CALCULATE_K( 5, buffer_wavefunction, buffer_reservoir );

                     // Stage 6 (t + c₆·h),   c₆ =  3.3333333333333333e-1
                     INTERMEDIATE_SUM_K( 5,
                                         Type::real( 3.7037037037037037e-2 ), // a₆₁
                                         Type::real( 0.0 ),                   // a₆₂
                                         Type::real( 0.0 ),                   // a₆₃
                                         Type::real( 1.7082860872947387e-1 ), // a₆₄
                                         Type::real( 1.2546768756682243e-1 )  // a₆₅
                     );
                     CALCULATE_K( 6, buffer_wavefunction, buffer_reservoir );

                     // Stage 7 (t + c₇·h),   c₇ =  2.5e-1
                     INTERMEDIATE_SUM_K( 6,
                                         Type::real( 3.7109375e-2 ),          // a₇₁
                                         Type::real( 0.0 ),                   // a₇₂
                                         Type::real( 0.0 ),                   // a₇₃
                                         Type::real( 1.7025221101954404e-1 ), // a₇₄
                                         Type::real( 6.0216538980455961e-2 ), // a₇₅
                                         Type::real( -1.7578125e-2 )          // a₇₆
                     );
                     CALCULATE_K( 7, buffer_wavefunction, buffer_reservoir );

                     // Stage 8 (t + c₈·h),   c₈ =  0.3076923076923077
                     INTERMEDIATE_SUM_K( 7,
                                         Type::real( 3.7092000118504793e-2 ),  // a₈₁
                                         Type::real( 0.0 ),                    // a₈₂
                                         Type::real( 0.0 ),                    // a₈₃
                                         Type::real( 1.7038392571223999e-1 ),  // a₈₄
                                         Type::real( 1.0726203044637328e-1 ),  // a₈₅
                                         Type::real( -1.5319437748624402e-2 ), // a₈₆
                                         Type::real( 8.2737891638140229e-3 )   // a₈₇
                     );
                     CALCULATE_K( 8, buffer_wavefunction, buffer_reservoir );

                     // Stage 9 (t + c₉·h),   c₉ =  0.6512820512820513
                     INTERMEDIATE_SUM_K( 8,
                                         Type::real( 6.2411095871607572e-1 ),  // a₉₁
                                         Type::real( 0.0 ),                    // a₉₂
                                         Type::real( 0.0 ),                    // a₉₃
                                         Type::real( -3.3608926294469413e+0 ), // a₉₄
                                         Type::real( -8.6821934684172601e-1 ), // a₉₅
                                         Type::real( 2.7592099699446708e+1 ),  // a₉₆
                                         Type::real( 2.0154067550477893e+1 ),  // a₉₇
                                         Type::real( -4.3489884181069959e+1 )  // a₉₈
                     );
                     CALCULATE_K( 9, buffer_wavefunction, buffer_reservoir );

                     // Stage 10 (t + c₁₀·h), c₁₀ =  0.6
                     INTERMEDIATE_SUM_K( 9,
                                         Type::real( 4.7766253643826437e-1 ),  // a₁₀,₁
                                         Type::real( 0.0 ),                    // a₁₀,₂
                                         Type::real( 0.0 ),                    // a₁₀,₃
                                         Type::real( -2.4881146199716676e+0 ), // a₁₀,₄
                                         Type::real( -5.9029082683684300e-1 ), // a₁₀,₅
                                         Type::real( 2.1230051448181194e+1 ),  // a₁₀,₆
                                         Type::real( 1.5279233632882424e+1 ),  // a₁₀,₇
                                         Type::real( -3.3288210968984863e+1 ), // a₁₀,₈
                                         Type::real( -2.0331201708508626e-2 )  // a₁₀,₉
                     );
                     CALCULATE_K( 10, buffer_wavefunction, buffer_reservoir );

                     // Stage 11 (t + c₁₁·h), c₁₁ =  6/7
                     INTERMEDIATE_SUM_K( 10,
                                         Type::real( -9.3714243008598733e-1 ), // a₁₁,₁
                                         Type::real( 0.0 ),                    // a₁₁,₂
                                         Type::real( 0.0 ),                    // a₁₁,₃
                                         Type::real( 5.1863724288440637e+0 ),  // a₁₁,₄
                                         Type::real( 1.0914373489967296e+0 ),  // a₁₁,₅
                                         Type::real( -8.1497870107469261e+0 ), // a₁₁,₆
                                         Type::real( -1.8520065659996960e+1 ), // a₁₁,₇
                                         Type::real( 2.2739487099350504e+1 ),  // a₁₁,₈
                                         Type::real( 2.4936055526796524e+0 ),  // a₁₁,₉
                                         Type::real( -3.0467644718982195e+0 )  // a₁₁,₁₀
                     );
                     CALCULATE_K( 11, buffer_wavefunction, buffer_reservoir );

                     // Stage 12 (t + c₁₂·h), c₁₂ =  1.0
                     INTERMEDIATE_SUM_K( 11,
                                         Type::real( 2.2733101475165382e+0 ),  // a₁₂,₁
                                         Type::real( 0.0 ),                    // a₁₂,₂
                                         Type::real( 0.0 ),                    // a₁₂,₃
                                         Type::real( -1.0534495466737250e+1 ), // a₁₂,₄
                                         Type::real( -2.0008720582248625e+0 ), // a₁₂,₅
                                         Type::real( -1.7958931863118799e+1 ), // a₁₂,₆
                                         Type::real( 2.7948884529419960e+1 ),  // a₁₂,₇
                                         Type::real( -2.8589982771350237e+0 ), // a₁₂,₈
                                         Type::real( -8.8728569335306295e+0 ), // a₁₂,₉
                                         Type::real( 1.2360567175794303e+1 ),  // a₁₂,₁₀
                                         Type::real( 6.4339274601576353e-1 )   // a₁₂,₁₁
                     );
                     CALCULATE_K( 12, buffer_wavefunction, buffer_reservoir );

                     // Final 8th‑order combination (b₁…b₁₂)
                     // b₂=b₃=b₄=b₅=0
                     FINAL_SUM_K( 12,
                                  Type::real( 5.4293734116568762e-2 ),  // b₁
                                  Type::real( 0.0 ),                    // b₂
                                  Type::real( 0.0 ),                    // b₃
                                  Type::real( 0.0 ),                    // b₄
                                  Type::real( 0.0 ),                    // b₅
                                  Type::real( 4.4503128927524089e+0 ),  // b₆
                                  Type::real( 1.8915178993145004e+0 ),  // b₇
                                  Type::real( -5.8012039600105848e+0 ), // b₈
                                  Type::real( 3.1116436695781989e-1 ),  // b₉
                                  Type::real( -1.5216094966251608e-1 ), // b₁₀
                                  Type::real( 2.0136540080403035e-1 ),  // b₁₁
                                  Type::real( 4.4710615727772591e-2 )   // b₁₂
                     );

    );
}

void PHOENIX::Solver::iterateVariableTimestepDOP853() {
    bool accept = false;
    do {
        SOLVER_SEQUENCE( false /*Capture CUDA Graph*/,
                         // ——— Dormand–Prince 8(5,3) (w) ———
                         // Coefficients taken verbatim from your dop853.f snippet.

                         // Stage 1 (t + 0)
                         CALCULATE_K( 1, wavefunction, reservoir );

                         // Stage 2 (t + c₂·h),   c₂ =  5.260015195876773e-2
                         INTERMEDIATE_SUM_K( 1, Type::real( 5.260015195876773e-2 ) ); CALCULATE_K( 2, buffer_wavefunction, buffer_reservoir );

                         // Stage 3 (t + c₃·h),   c₃ =  7.8900227938151598e-2
                         INTERMEDIATE_SUM_K( 2,
                                             Type::real( 1.9725056984537899e-2 ), // a₃₁
                                             Type::real( 5.9175170953613698e-2 )  // a₃₂
                         );
                         CALCULATE_K( 3, buffer_wavefunction, buffer_reservoir );

                         // Stage 4 (t + c₄·h),   c₄ =  1.1835034190722739e-1
                         INTERMEDIATE_SUM_K( 3,
                                             Type::real( 2.9587585476806849e-2 ), // a₄₁
                                             Type::real( 0.0 ),                   // a₄₂
                                             Type::real( 8.8762756430420548e-2 )  // a₄₃
                         );
                         CALCULATE_K( 4, buffer_wavefunction, buffer_reservoir );

                         // Stage 5 (t + c₅·h),   c₅ =  2.8164965809277260e-1
                         INTERMEDIATE_SUM_K( 4,
                                             Type::real( 2.4136513415926669e-1 ),  // a₅₁
                                             Type::real( 0.0 ),                    // a₅₂
                                             Type::real( -8.8454947932828610e-1 ), // a₅₃
                                             Type::real( 9.2483400326179200e-1 )   // a₅₄
                         );
                         CALCULATE_K( 5, buffer_wavefunction, buffer_reservoir );

                         // Stage 6 (t + c₆·h),   c₆ =  3.3333333333333333e-1
                         INTERMEDIATE_SUM_K( 5,
                                             Type::real( 3.7037037037037037e-2 ), // a₆₁
                                             Type::real( 0.0 ),                   // a₆₂
                                             Type::real( 0.0 ),                   // a₆₃
                                             Type::real( 1.7082860872947387e-1 ), // a₆₄
                                             Type::real( 1.2546768756682243e-1 )  // a₆₅
                         );
                         CALCULATE_K( 6, buffer_wavefunction, buffer_reservoir );

                         // Stage 7 (t + c₇·h),   c₇ =  2.5e-1
                         INTERMEDIATE_SUM_K( 6,
                                             Type::real( 3.7109375e-2 ),          // a₇₁
                                             Type::real( 0.0 ),                   // a₇₂
                                             Type::real( 0.0 ),                   // a₇₃
                                             Type::real( 1.7025221101954404e-1 ), // a₇₄
                                             Type::real( 6.0216538980455961e-2 ), // a₇₅
                                             Type::real( -1.7578125e-2 )          // a₇₆
                         );
                         CALCULATE_K( 7, buffer_wavefunction, buffer_reservoir );

                         // Stage 8 (t + c₈·h),   c₈ =  0.3076923076923077
                         INTERMEDIATE_SUM_K( 7,
                                             Type::real( 3.7092000118504793e-2 ),  // a₈₁
                                             Type::real( 0.0 ),                    // a₈₂
                                             Type::real( 0.0 ),                    // a₈₃
                                             Type::real( 1.7038392571223999e-1 ),  // a₈₄
                                             Type::real( 1.0726203044637328e-1 ),  // a₈₅
                                             Type::real( -1.5319437748624402e-2 ), // a₈₆
                                             Type::real( 8.2737891638140229e-3 )   // a₈₇
                         );
                         CALCULATE_K( 8, buffer_wavefunction, buffer_reservoir );

                         // Stage 9 (t + c₉·h),   c₉ =  0.6512820512820513
                         INTERMEDIATE_SUM_K( 8,
                                             Type::real( 6.2411095871607572e-1 ),  // a₉₁
                                             Type::real( 0.0 ),                    // a₉₂
                                             Type::real( 0.0 ),                    // a₉₃
                                             Type::real( -3.3608926294469413e+0 ), // a₉₄
                                             Type::real( -8.6821934684172601e-1 ), // a₉₅
                                             Type::real( 2.7592099699446708e+1 ),  // a₉₆
                                             Type::real( 2.0154067550477893e+1 ),  // a₉₇
                                             Type::real( -4.3489884181069959e+1 )  // a₉₈
                         );
                         CALCULATE_K( 9, buffer_wavefunction, buffer_reservoir );

                         // Stage 10 (t + c₁₀·h), c₁₀ =  0.6
                         INTERMEDIATE_SUM_K( 9,
                                             Type::real( 4.7766253643826437e-1 ),  // a₁₀,₁
                                             Type::real( 0.0 ),                    // a₁₀,₂
                                             Type::real( 0.0 ),                    // a₁₀,₃
                                             Type::real( -2.4881146199716676e+0 ), // a₁₀,₄
                                             Type::real( -5.9029082683684300e-1 ), // a₁₀,₅
                                             Type::real( 2.1230051448181194e+1 ),  // a₁₀,₆
                                             Type::real( 1.5279233632882424e+1 ),  // a₁₀,₇
                                             Type::real( -3.3288210968984863e+1 ), // a₁₀,₈
                                             Type::real( -2.0331201708508626e-2 )  // a₁₀,₉
                         );
                         CALCULATE_K( 10, buffer_wavefunction, buffer_reservoir );

                         // Stage 11 (t + c₁₁·h), c₁₁ =  6/7
                         INTERMEDIATE_SUM_K( 10,
                                             Type::real( -9.3714243008598733e-1 ), // a₁₁,₁
                                             Type::real( 0.0 ),                    // a₁₁,₂
                                             Type::real( 0.0 ),                    // a₁₁,₃
                                             Type::real( 5.1863724288440637e+0 ),  // a₁₁,₄
                                             Type::real( 1.0914373489967296e+0 ),  // a₁₁,₅
                                             Type::real( -8.1497870107469261e+0 ), // a₁₁,₆
                                             Type::real( -1.8520065659996960e+1 ), // a₁₁,₇
                                             Type::real( 2.2739487099350504e+1 ),  // a₁₁,₈
                                             Type::real( 2.4936055526796524e+0 ),  // a₁₁,₉
                                             Type::real( -3.0467644718982195e+0 )  // a₁₁,₁₀
                         );
                         CALCULATE_K( 11, buffer_wavefunction, buffer_reservoir );

                         // Stage 12 (t + c₁₂·h), c₁₂ =  1.0
                         INTERMEDIATE_SUM_K( 11,
                                             Type::real( 2.2733101475165382e+0 ),  // a₁₂,₁
                                             Type::real( 0.0 ),                    // a₁₂,₂
                                             Type::real( 0.0 ),                    // a₁₂,₃
                                             Type::real( -1.0534495466737250e+1 ), // a₁₂,₄
                                             Type::real( -2.0008720582248625e+0 ), // a₁₂,₅
                                             Type::real( -1.7958931863118799e+1 ), // a₁₂,₆
                                             Type::real( 2.7948884529419960e+1 ),  // a₁₂,₇
                                             Type::real( -2.8589982771350237e+0 ), // a₁₂,₈
                                             Type::real( -8.8728569335306295e+0 ), // a₁₂,₉
                                             Type::real( 1.2360567175794303e+1 ),  // a₁₂,₁₀
                                             Type::real( 6.4339274601576353e-1 )   // a₁₂,₁₁
                         );
                         CALCULATE_K( 12, buffer_wavefunction, buffer_reservoir );

                         // Final 8th‑order combination (b₁…b₁₂)
                         // b₂=b₃=b₄=b₅=0
                         INTERMEDIATE_SUM_K( 12,
                                             Type::real( 5.4293734116568762e-2 ),  // b₁
                                             Type::real( 0.0 ),                    // b₂
                                             Type::real( 0.0 ),                    // b₃
                                             Type::real( 0.0 ),                    // b₄
                                             Type::real( 0.0 ),                    // b₅
                                             Type::real( 4.4503128927524089e+0 ),  // b₆
                                             Type::real( 1.8915178993145004e+0 ),  // b₇
                                             Type::real( -5.8012039600105848e+0 ), // b₈
                                             Type::real( 3.1116436695781989e-1 ),  // b₉
                                             Type::real( -1.5216094966251608e-1 ), // b₁₀
                                             Type::real( 2.0136540080403035e-1 ),  // b₁₁
                                             Type::real( 4.4710615727772591e-2 )   // b₁₂
                         );

                         ERROR_K( 12,
                                  Type::real( 4.1173689122373888e-2 ),  // = 0.05429373411656876 − 0.01312004499419488
                                  Type::real( 0.0 ),                    // b2*
                                  Type::real( 0.0 ),                    // b3*
                                  Type::real( 0.0 ),                    // b4*
                                  Type::real( 0.0 ),                    // b5*
                                  Type::real( 5.6754693391286133e+0 ),  // = 4.450312892752409 + 1.2251564463762044
                                  Type::real( 2.3872768489717506e+0 ),  // = 1.8915178993145004 + 0.4957589496572502
                                  Type::real( -7.4655811424655713e+0 ), // = −5.801203960010585 − 1.6643771824549865
                                  Type::real( 6.6149321570779360e-1 ),  // = 0.3111643669578194 + 0.3503288487499737
                                  Type::real( -4.8634006837553356e-1 ), // = −0.1521609496625161 − 0.3341791187130175
                                  Type::real( 1.1944219431891464e-1 ),  // = 0.2013654008040304 − 0.0819232064851157
                                  Type::real( 6.7065923591658886e-2 )   // = 0.04471061572777259 + 0.02235530786388630
                         );

        );
        //auto [min, max] = matrix.buffer_wavefunction_plus.extrema();
        auto msum = matrix.buffer_wavefunction_plus.transformReduce( Type::complex( 0.0 ), CUDAMatrix<Type::complex>::transform_abs2(), CUDAMatrix<Type::complex>::transform_sum() );
        Type::real normalization_factor = CUDA::sqrt( CUDA::real( msum ) );
        Type::real integrated_error = std::sqrt( matrix.rk_error.sum() );

        Type::real final_error = std::abs( integrated_error / normalization_factor );
        Type::real dh_arg = system.tolerance / 2.0 / CUDA::max( std::numeric_limits<Type::real>::min(), final_error );
        Type::real dh = std::pow<Type::real>( dh_arg, Type::real( 0.16 ) );

        if ( std::isnan( dh ) ) {
            dh = 0.9;
            final_error = std::numeric_limits<Type::real>::max();
        }
        if ( std::isnan( final_error ) ) {
            dh = 0.9;
            final_error = std::numeric_limits<Type::real>::max();
        }

        //  Set new timestep
        Type::real new_dt = std::min( system.p.dt * dh, system.dt_max );
        //if ( dh < 1.0 )
        //new_dt = std::max( system.p.dt - system.dt_min * std::floor( 1.0 / dh ), system.dt_min );
        //else
        //new_dt = std::min( system.p.dt + system.dt_min * std::floor( dh ), system.dt_max );
        if ( new_dt < system.dt_min ) {
            new_dt = system.dt_min;
            accept = true;
        }
        system.p.dt = new_dt;

        updateKernelTime();

        if ( final_error < system.tolerance ) {
            accept = true;
            matrix.wavefunction_plus.swap( matrix.buffer_wavefunction_plus );
            matrix.reservoir_plus.swap( matrix.buffer_reservoir_plus );
            if ( system.use_twin_mode ) {
                matrix.wavefunction_minus.swap( matrix.buffer_wavefunction_minus );
                matrix.reservoir_minus.swap( matrix.buffer_reservoir_minus );
            }
            //std::cout << "ACCEPTED " << final_error << ", norm = " << normalization_factor << ", dh = " << dh << " --> new dt = " << new_dt << " (old: " << system.p.dt << ")" << std::endl;
        }

        //std::cout << final_error << ", norm = " << normalization_factor << ", dh = " << dh << " --> new dt = " << new_dt << " (old: " << system.p.dt << ")" << std::endl;
    } while ( !accept );
}