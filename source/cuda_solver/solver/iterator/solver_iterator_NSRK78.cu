#include "hip/hip_runtime.h"
#pragma once
#include <cmath>
#include <omp.h>

// Include Cuda Kernel headers
#include "cuda/typedef.cuh"
#include "kernel/kernel_compute.cuh"
#include "kernel/kernel_summation.cuh"
#include "kernel/kernel_halo.cuh"
#include "system/system_parameters.hpp"
#include "cuda/cuda_matrix.cuh"
#include "solver/gpu_solver.hpp"
#include "misc/commandline_io.hpp"

void PHOENIX::Solver::iterateFixedTimestepNSRK78() {
    SOLVER_SEQUENCE( true /* capture CUDA graph */,
                     // ——— Nullspace‑efficient RK 13(8:7) ———
                     // Stage 1 (t + 0)
                     CALCULATE_K( 1, Type::real( 0.0 ), wavefunction, reservoir );

                     // Stage 2 (t + c₂·h), c₂ = 1/1000
                     INTERMEDIATE_SUM_K( 1, Type::real( 1.0 / 1000.0 ) );

                     CALCULATE_K( 2, Type::real( 0.0010000000 ), buffer_wavefunction, buffer_reservoir );

                     // Stage 3 (t + c₃·h), c₃ = 1/9
                     INTERMEDIATE_SUM_K( 2,
                                         Type::real( -491.0 / 81.0 ), // a₃₁
                                         Type::real( 500.0 / 81.0 )   // a₃₂
                     );
                     CALCULATE_K( 3, Type::real( 0.1111111111 ), buffer_wavefunction, buffer_reservoir );

                     // Stage 4 (t + c₄·h), c₄ = 1/6
                     INTERMEDIATE_SUM_K( 3,
                                         Type::real( 1.0 / 24.0 ), // a₄₁
                                         Type::real( 0.0 ),        // a₄₂
                                         Type::real( 1.0 / 8.0 )   // a₄₃
                     );
                     CALCULATE_K( 4, Type::real( 0.1666666667 ), buffer_wavefunction, buffer_reservoir );

                     // Stage 5 (t + c₅·h), c₅ = 5/12
                     INTERMEDIATE_SUM_K( 4,
                                         Type::real( 5.0 / 12.0 ), // a₅₁
                                         Type::real( 0.0 ),
                                         Type::real( -25.0 / 16.0 ), // a₅₃
                                         Type::real( 25.0 / 16.0 )   // a₅₄
                     );
                     CALCULATE_K( 5, Type::real( 0.4166666667 ), buffer_wavefunction, buffer_reservoir );

                     // Stage 6 (t + c₆·h), c₆ = 1/2
                     INTERMEDIATE_SUM_K( 5,
                                         Type::real( 1.0 / 20.0 ), // a₆₁
                                         Type::real( 0.0 ), Type::real( 0.0 ),
                                         Type::real( 1.0 / 4.0 ), // a₆₄
                                         Type::real( 1.0 / 5.0 )  // a₆₅
                     );
                     CALCULATE_K( 6, Type::real( 0.5 ), buffer_wavefunction, buffer_reservoir );

                     // Stage 7 (t + c₇·h), c₇ = 5/6
                     INTERMEDIATE_SUM_K( 6,
                                         Type::real( -43.0 / 180.0 ), // a₇₁
                                         Type::real( 0.0 ), Type::real( 0.0 ),
                                         Type::real( 127.0 / 108.0 ),  // a₇₄
                                         Type::real( -329.0 / 135.0 ), // a₇₅
                                         Type::real( 7.0 / 3.0 )       // a₇₆
                     );
                     CALCULATE_K( 7, Type::real( 0.8333333333 ), buffer_wavefunction, buffer_reservoir );

                     // Stage 8 (t + c₈·h), c₈ = 1/6
                     INTERMEDIATE_SUM_K( 7,
                                         Type::real( 27931.0 / 240300.0 ), // a₈₁
                                         Type::real( 0.0 ), Type::real( 0.0 ),
                                         Type::real( -631.0 / 16020.0 ),  // a₈₄
                                         Type::real( 2459.0 / 6675.0 ),   // a₈₅
                                         Type::real( -3572.0 / 12015.0 ), // a₈₆
                                         Type::real( 5.0 / 267.0 )        // a₈₇
                     );
                     CALCULATE_K( 8, Type::real( 0.1666666667 ), buffer_wavefunction, buffer_reservoir );

                     // Stage 9 (t + c₉·h), c₉ = 2/3
                     INTERMEDIATE_SUM_K( 8,
                                         Type::real( 26114.0 / 12015.0 ), // a₉₁
                                         Type::real( 0.0 ), Type::real( 0.0 ),
                                         Type::real( -7480.0 / 801.0 ),   // a₉₄
                                         Type::real( 67264.0 / 4005.0 ),  // a₉₅
                                         Type::real( -30640.0 / 2403.0 ), // a₉₆
                                         Type::real( 1051.0 / 1335.0 ),   // a₉₇
                                         Type::real( 3.0 )                // a₉₈
                     );
                     CALCULATE_K( 9, Type::real( 0.6666666667 ), buffer_wavefunction, buffer_reservoir );

                     // Stage 10 (t + c₁₀·h), c₁₀ = 19/25
                     INTERMEDIATE_SUM_K( 9,
                                         Type::real( 33096587331.0 / 17382812500.0 ), // a₁₀,₁
                                         Type::real( 0.0 ), Type::real( 0.0 ),
                                         Type::real( -1828977848.0 / 173828125.0 ),  // a₁₀,₄
                                         Type::real( 62801809904.0 / 4345703125.0 ), // a₁₀,₅
                                         Type::real( -9389764774.0 / 869140625.0 ),  // a₁₀,₆
                                         Type::real( 6380757669.0 / 8691406250.0 ),  // a₁₀,₇
                                         Type::real( 98417891.0 / 19531250.0 ),      // a₁₀,₈
                                         Type::real( -1692691.0 / 39062500.0 )       // a₁₀,₉
                     );
                     CALCULATE_K( 10, Type::real( 0.76 ), buffer_wavefunction, buffer_reservoir );

                     // Stage 11 (t + c₁₁·h), c₁₁ = 21/25
                     INTERMEDIATE_SUM_K( 10,
                                         Type::real( -1456295425347.0 / 2642187500000.0 ), // a₁₁,₁
                                         Type::real( 0.0 ), Type::real( 0.0 ),
                                         Type::real( 110740056.0 / 34765625.0 ),          // a₁₁,₄
                                         Type::real( -21221682384.0 / 4345703125.0 ),     // a₁₁,₅
                                         Type::real( 58859060169.0 / 13906250000.0 ),     // a₁₁,₆
                                         Type::real( -177381525069.0 / 1529687500000.0 ), // a₁₁,₇
                                         Type::real( -28942485159.0 / 27812500000.0 ),    // a₁₁,₈
                                         Type::real( -1272297.0 / 312500000.0 ),          // a₁₁,₉
                                         Type::real( 5151.0 / 297616.0 )                  // a₁₁,₁₀
                     );
                     CALCULATE_K( 11, Type::real( 0.84 ), buffer_wavefunction, buffer_reservoir );

                     // Stage 12 (t + c₁₂·h), c₁₂ = 1
                     INTERMEDIATE_SUM_K( 11,
                                         Type::real( 844300798.0 / 137013275.0 ), // a₁₂,₁
                                         Type::real( 0.0 ), Type::real( 0.0 ),
                                         Type::real( -9997568.0 / 206035.0 ),       // a₁₂,₄
                                         Type::real( 49636624.0 / 1030175.0 ),      // a₁₂,₅
                                         Type::real( -3358834871.0 / 91067470.0 ),  // a₁₂,₆
                                         Type::real( -40456983.0 / 1813108.0 ),     // a₁₂,₇
                                         Type::real( 495817135.0 / 16647628.0 ),    // a₁₂,₈
                                         Type::real( -149375.0 / 84266.0 ),         // a₁₂,₉
                                         Type::real( 7470703125.0 / 1567431866.0 ), // a₁₂,₁₀
                                         Type::real( 1562500000.0 / 72342361.0 )    // a₁₂,₁₁
                     );

                     CALCULATE_K( 12, Type::real( 1.0 ), buffer_wavefunction, buffer_reservoir );

                     // Stage 13 (t + c₁₃·h), c₁₃ = 1
                     INTERMEDIATE_SUM_K( 12,
                                         Type::real( -26225423.0 / 37371100.0 ), // a₁₃,₁
                                         Type::real( 0.0 ), Type::real( 0.0 ),
                                         Type::real( 807744.0 / 98345.0 ),        // a₁₃,₄
                                         Type::real( -205584.0 / 37825.0 ),       // a₁₃,₅
                                         Type::real( 5882202.0 / 1278485.0 ),     // a₁₃,₆
                                         Type::real( -84543.0 / 432718.0 ),       // a₁₃,₇
                                         Type::real( -223415.0 / 39338.0 ),       // a₁₃,₈
                                         Type::real( -3625.0 / 6188.0 ),          // a₁₃,₉
                                         Type::real( 292968750.0 / 374084711.0 ), // a₁₃,₁₀
                                         Type::real( 0.0 ),                       // a₁₃,₁₁
                                         Type::real( 0.0 )                        // a₁₃,₁₂
                     );

                     CALCULATE_K( 13, Type::real( 1.0 ), buffer_wavefunction, buffer_reservoir );

                     // High‑order (8th) solution
                     FINAL_SUM_K( 13,
                                  Type::real( 4241.0 / 88200.0 ), // b₁
                                  Type::real( 0.0 ), Type::real( 0.0 ), Type::real( 0.0 ), Type::real( 0.0 ), Type::real( 9946.0 / 23205.0 ), Type::real( -10449.0 / 1925.0 ), Type::real( 415449.0 / 1573075.0 ), Type::real( -2025.0 / 5096.0 ), Type::real( 48828125.0 / 44900856.0 ), Type::real( 48828125.0 / 9843561.0 ), Type::real( 463.0 / 12600.0 ), Type::real( 0.0 ) ); );
}

void PHOENIX::Solver::iterateVariableTimestepNSRK78() {
    bool accept = false;
    do {
        SOLVER_SEQUENCE( false /* no graph capture */,
                         // ——— Nullspace‑efficient RK 13(8:7) ———
                         // Stage 1 (t + 0)
                         CALCULATE_K( 1, Type::real( 0.0 ), wavefunction, reservoir );

                         // Stage 2 (t + c₂·h), c₂ = 1/1000
                         INTERMEDIATE_SUM_K( 1, Type::real( 1.0 / 1000.0 ) );

                         CALCULATE_K( 2, Type::real( 0.001 ), buffer_wavefunction, buffer_reservoir );

                         // Stage 3 (t + c₃·h), c₃ = 1/9
                         INTERMEDIATE_SUM_K( 2,
                                             Type::real( -491.0 / 81.0 ), // a₃₁
                                             Type::real( 500.0 / 81.0 )   // a₃₂
                         );
                         CALCULATE_K( 3, Type::real( 0.1111111111 ), buffer_wavefunction, buffer_reservoir );

                         // Stage 4 (t + c₄·h), c₄ = 1/6
                         INTERMEDIATE_SUM_K( 3,
                                             Type::real( 1.0 / 24.0 ), // a₄₁
                                             Type::real( 0.0 ),        // a₄₂
                                             Type::real( 1.0 / 8.0 )   // a₄₃
                         );
                         CALCULATE_K( 4, Type::real( 0.1666666667 ), buffer_wavefunction, buffer_reservoir );

                         // Stage 5 (t + c₅·h), c₅ = 5/12
                         INTERMEDIATE_SUM_K( 4,
                                             Type::real( 5.0 / 12.0 ), // a₅₁
                                             Type::real( 0.0 ),
                                             Type::real( -25.0 / 16.0 ), // a₅₃
                                             Type::real( 25.0 / 16.0 )   // a₅₄
                         );
                         CALCULATE_K( 5, Type::real( 0.4166666667 ), buffer_wavefunction, buffer_reservoir );

                         // Stage 6 (t + c₆·h), c₆ = 1/2
                         INTERMEDIATE_SUM_K( 5,
                                             Type::real( 1.0 / 20.0 ), // a₆₁
                                             Type::real( 0.0 ), Type::real( 0.0 ),
                                             Type::real( 1.0 / 4.0 ), // a₆₄
                                             Type::real( 1.0 / 5.0 )  // a₆₅
                         );
                         CALCULATE_K( 6, Type::real( 0.5 ), buffer_wavefunction, buffer_reservoir );

                         // Stage 7 (t + c₇·h), c₇ = 5/6
                         INTERMEDIATE_SUM_K( 6,
                                             Type::real( -43.0 / 180.0 ), // a₇₁
                                             Type::real( 0.0 ), Type::real( 0.0 ),
                                             Type::real( 127.0 / 108.0 ),  // a₇₄
                                             Type::real( -329.0 / 135.0 ), // a₇₅
                                             Type::real( 7.0 / 3.0 )       // a₇₆
                         );
                         CALCULATE_K( 7, Type::real( 0.8333333333 ), buffer_wavefunction, buffer_reservoir );

                         // Stage 8 (t + c₈·h), c₈ = 1/6
                         INTERMEDIATE_SUM_K( 7,
                                             Type::real( 27931.0 / 240300.0 ), // a₈₁
                                             Type::real( 0.0 ), Type::real( 0.0 ),
                                             Type::real( -631.0 / 16020.0 ),  // a₈₄
                                             Type::real( 2459.0 / 6675.0 ),   // a₈₅
                                             Type::real( -3572.0 / 12015.0 ), // a₈₆
                                             Type::real( 5.0 / 267.0 )        // a₈₇
                         );
                         CALCULATE_K( 8, Type::real( 0.1666666667 ), buffer_wavefunction, buffer_reservoir );

                         // Stage 9 (t + c₉·h), c₉ = 2/3
                         INTERMEDIATE_SUM_K( 8,
                                             Type::real( 26114.0 / 12015.0 ), // a₉₁
                                             Type::real( 0.0 ), Type::real( 0.0 ),
                                             Type::real( -7480.0 / 801.0 ),   // a₉₄
                                             Type::real( 67264.0 / 4005.0 ),  // a₉₅
                                             Type::real( -30640.0 / 2403.0 ), // a₉₆
                                             Type::real( 1051.0 / 1335.0 ),   // a₉₇
                                             Type::real( 3.0 )                // a₉₈
                         );
                         CALCULATE_K( 9, Type::real( 0.6666666667 ), buffer_wavefunction, buffer_reservoir );

                         // Stage 10 (t + c₁₀·h), c₁₀ = 19/25
                         INTERMEDIATE_SUM_K( 9,
                                             Type::real( 33096587331.0 / 17382812500.0 ), // a₁₀,₁
                                             Type::real( 0.0 ), Type::real( 0.0 ),
                                             Type::real( -1828977848.0 / 173828125.0 ),  // a₁₀,₄
                                             Type::real( 62801809904.0 / 4345703125.0 ), // a₁₀,₅
                                             Type::real( -9389764774.0 / 869140625.0 ),  // a₁₀,₆
                                             Type::real( 6380757669.0 / 8691406250.0 ),  // a₁₀,₇
                                             Type::real( 98417891.0 / 19531250.0 ),      // a₁₀,₈
                                             Type::real( -1692691.0 / 39062500.0 )       // a₁₀,₉
                         );
                         CALCULATE_K( 10, Type::real( 0.76 ), buffer_wavefunction, buffer_reservoir );

                         // Stage 11 (t + c₁₁·h), c₁₁ = 21/25
                         INTERMEDIATE_SUM_K( 10,
                                             Type::real( -1456295425347.0 / 2642187500000.0 ), // a₁₁,₁
                                             Type::real( 0.0 ), Type::real( 0.0 ),
                                             Type::real( 110740056.0 / 34765625.0 ),          // a₁₁,₄
                                             Type::real( -21221682384.0 / 4345703125.0 ),     // a₁₁,₅
                                             Type::real( 58859060169.0 / 13906250000.0 ),     // a₁₁,₆
                                             Type::real( -177381525069.0 / 1529687500000.0 ), // a₁₁,₇
                                             Type::real( -28942485159.0 / 27812500000.0 ),    // a₁₁,₈
                                             Type::real( -1272297.0 / 312500000.0 ),          // a₁₁,₉
                                             Type::real( 5151.0 / 297616.0 )                  // a₁₁,₁₀
                         );
                         CALCULATE_K( 11, Type::real( 0.84 ), buffer_wavefunction, buffer_reservoir );

                         // Stage 12 (t + c₁₂·h), c₁₂ = 1
                         INTERMEDIATE_SUM_K( 11,
                                             Type::real( 844300798.0 / 137013275.0 ), // a₁₂,₁
                                             Type::real( 0.0 ), Type::real( 0.0 ),
                                             Type::real( -9997568.0 / 206035.0 ),       // a₁₂,₄
                                             Type::real( 49636624.0 / 1030175.0 ),      // a₁₂,₅
                                             Type::real( -3358834871.0 / 91067470.0 ),  // a₁₂,₆
                                             Type::real( -40456983.0 / 1813108.0 ),     // a₁₂,₇
                                             Type::real( 495817135.0 / 16647628.0 ),    // a₁₂,₈
                                             Type::real( -149375.0 / 84266.0 ),         // a₁₂,₉
                                             Type::real( 7470703125.0 / 1567431866.0 ), // a₁₂,₁₀
                                             Type::real( 1562500000.0 / 72342361.0 )    // a₁₂,₁₁
                         );
                         CALCULATE_K( 12, Type::real( 1.0 ), buffer_wavefunction, buffer_reservoir );

                         // Stage 13 (t + c₁₃·h), c₁₃ = 1
                         INTERMEDIATE_SUM_K( 12,
                                             Type::real( -26225423.0 / 37371100.0 ), // a₁₃,₁
                                             Type::real( 0.0 ), Type::real( 0.0 ),
                                             Type::real( 807744.0 / 98345.0 ),        // a₁₃,₄
                                             Type::real( -205584.0 / 37825.0 ),       // a₁₃,₅
                                             Type::real( 5882202.0 / 1278485.0 ),     // a₁₃,₆
                                             Type::real( -84543.0 / 432718.0 ),       // a₁₃,₇
                                             Type::real( -223415.0 / 39338.0 ),       // a₁₃,₈
                                             Type::real( -3625.0 / 6188.0 ),          // a₁₃,₉
                                             Type::real( 292968750.0 / 374084711.0 ), // a₁₃,₁₀
                                             Type::real( 0.0 ),                       // a₁₃,₁₁
                                             Type::real( 0.0 )                        // a₁₃,₁₂
                         );
                         CALCULATE_K( 13, Type::real( 1.0 ), buffer_wavefunction, buffer_reservoir );

                         // High‑order (8th) solution
                         INTERMEDIATE_SUM_K( 13,
                                             Type::real( 4241.0 / 88200.0 ), // b₁
                                             Type::real( 0.0 ), Type::real( 0.0 ), Type::real( 0.0 ), Type::real( 0.0 ), Type::real( 9946.0 / 23205.0 ), Type::real( -10449.0 / 1925.0 ), Type::real( 415449.0 / 1573075.0 ), Type::real( -2025.0 / 5096.0 ), Type::real( 48828125.0 / 44900856.0 ), Type::real( 48828125.0 / 9843561.0 ), Type::real( 463.0 / 12600.0 ), Type::real( 0.0 ) );

                         // Embedded (7th) – error=high−low
                         ERROR_K( 13,
                                  Type::real( 4241.0 / 88200.0 - 3799.0 / 79800.0 ), // b₁−bh₁
                                  Type::real( 0.0 ), Type::real( 0.0 ), Type::real( 0.0 ), Type::real( 0.0 ), Type::real( 9946.0 / 23205.0 - 538.0 / 1365.0 ), Type::real( -10449.0 / 1925.0 - 351.0 / 1925.0 ), Type::real( 415449.0 / 1573075.0 - 4149.0 / 15575.0 ), Type::real( -2025.0 / 5096.0 - ( -45.0 / 392.0 ) ), Type::real( 48828125.0 / 44900856.0 - 48828125.0 / 284372088.0 ), Type::real( 48828125.0 / 9843561.0 - 0.0 ), Type::real( 463.0 / 12600.0 - 0.0 ), Type::real( 0.0 - 221.0 / 4200.0 ) );

        );

        // compute error norms exactly as in DOP853:
        auto msum = matrix.buffer_wavefunction_plus.transformReduce( Type::complex( 0.0 ), CUDAMatrix<Type::complex>::transform_abs2(), CUDAMatrix<Type::complex>::transform_sum() );
        Type::real normalization_factor = CUDA::sqrt( CUDA::real( msum ) );
        Type::real integrated_error = std::sqrt( matrix.rk_error.sum() );
        Type::real final_error = std::abs( integrated_error / normalization_factor );

        // stepsize controller: p=7 ⇒ exponent=1/8
        Type::real dh_arg = system.tolerance / 2.0 / CUDA::max( std::numeric_limits<Type::real>::min(), final_error );
        Type::real dh = std::pow( dh_arg, Type::real( 1.0 / 8.0 ) );

        if ( std::isnan( dh ) || std::isnan( final_error ) ) {
            dh = Type::real( 0.9 );
            final_error = std::numeric_limits<Type::real>::max();
        }

        // update dt within [dt_min,dt_max]
        Type::real new_dt = std::min( system.p.dt * dh, system.dt_max );
        if ( new_dt < system.dt_min ) {
            new_dt = system.dt_min;
            accept = true; // force accept at minimum dt
        }
        system.p.dt = new_dt;
        updateKernelTime();

        if ( final_error < system.tolerance ) {
            accept = true;
            // swap “plus” buffers into the solution
            matrix.wavefunction_plus.swap( matrix.buffer_wavefunction_plus );
            matrix.reservoir_plus.swap( matrix.buffer_reservoir_plus );
            if ( system.use_twin_mode ) {
                matrix.wavefunction_minus.swap( matrix.buffer_wavefunction_minus );
                matrix.reservoir_minus.swap( matrix.buffer_reservoir_minus );
            }
        }
    } while ( !accept );
}
