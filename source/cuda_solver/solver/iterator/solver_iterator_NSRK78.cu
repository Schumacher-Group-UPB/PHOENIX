#include "hip/hip_runtime.h"
#pragma once
#include <cmath>
#include <omp.h>

// Include Cuda Kernel headers
#include "cuda/typedef.cuh"
#include "kernel/kernel_compute.cuh"
#include "kernel/kernel_summation.cuh"
#include "kernel/kernel_halo.cuh"
#include "system/system_parameters.hpp"
#include "cuda/cuda_matrix.cuh"
#include "solver/gpu_solver.hpp"
#include "misc/commandline_io.hpp"

void PHOENIX::Solver::iterateFixedTimestepNSRK78() {
    SOLVER_SEQUENCE( true /* capture CUDA graph */,

                     CALCULATE_K( 1, Type::real( 0.0 ), wavefunction, reservoir );

                     INTERMEDIATE_SUM_K( 1, Type::real( 1.0 / 1000.0 ) );

                     CALCULATE_K( 2, Type::real( 0.001 ), buffer_wavefunction, buffer_reservoir );

                     INTERMEDIATE_SUM_K( 2, Type::real( -491.0 / 81.0 ), Type::real( 500.0 / 81.0 ) );

                     CALCULATE_K( 3, Type::real( 0.1111111111 ), buffer_wavefunction, buffer_reservoir );

                     INTERMEDIATE_SUM_K( 3, Type::real( 1.0 / 24.0 ), Type::real( 0.0 ), Type::real( 1.0 / 8.0 ) );

                     CALCULATE_K( 4, Type::real( 0.1666666667 ), buffer_wavefunction, buffer_reservoir );

                     INTERMEDIATE_SUM_K( 4, Type::real( 5.0 / 12.0 ), Type::real( 0.0 ), Type::real( -25.0 / 16.0 ), Type::real( 25.0 / 16.0 ) );

                     CALCULATE_K( 5, Type::real( 0.4166666667 ), buffer_wavefunction, buffer_reservoir );

                     INTERMEDIATE_SUM_K( 5, Type::real( 1.0 / 20.0 ), Type::real( 0.0 ), Type::real( 0.0 ), Type::real( 1.0 / 4.0 ), Type::real( 1.0 / 5.0 ) );

                     CALCULATE_K( 6, Type::real( 0.5 ), buffer_wavefunction, buffer_reservoir );

                     INTERMEDIATE_SUM_K( 6, Type::real( -43.0 / 180.0 ), Type::real( 0.0 ), Type::real( 0.0 ), Type::real( 127.0 / 108.0 ), Type::real( -329.0 / 135.0 ), Type::real( 7.0 / 3.0 ) );

                     CALCULATE_K( 7, Type::real( 0.8333333333 ), buffer_wavefunction, buffer_reservoir );

                     INTERMEDIATE_SUM_K( 7, Type::real( 27931.0 / 240300.0 ), Type::real( 0.0 ), Type::real( 0.0 ), Type::real( -631.0 / 16020.0 ), Type::real( 2459.0 / 6675.0 ), Type::real( -3572.0 / 12015.0 ), Type::real( 5.0 / 267.0 ) );

                     CALCULATE_K( 8, Type::real( 0.1666666667 ), buffer_wavefunction, buffer_reservoir );

                     INTERMEDIATE_SUM_K( 8, Type::real( 26114.0 / 12015.0 ), Type::real( 0.0 ), Type::real( 0.0 ), Type::real( -7480.0 / 801.0 ), Type::real( 67264.0 / 4005.0 ), Type::real( -30640.0 / 2403.0 ), Type::real( 1051.0 / 1335.0 ), Type::real( 3.0 ) );

                     CALCULATE_K( 9, Type::real( 0.6666666667 ), buffer_wavefunction, buffer_reservoir );

                     INTERMEDIATE_SUM_K( 9, Type::real( 33096587331.0 / 17382812500.0 ), Type::real( 0.0 ), Type::real( 0.0 ), Type::real( -1828977848.0 / 173828125.0 ), Type::real( 62801809904.0 / 4345703125.0 ), Type::real( -9389764774.0 / 869140625.0 ), Type::real( 6380757669.0 / 8691406250.0 ), Type::real( 98417891.0 / 19531250.0 ), Type::real( -1692691.0 / 39062500.0 ) );

                     CALCULATE_K( 10, Type::real( 0.76 ), buffer_wavefunction, buffer_reservoir );

                     INTERMEDIATE_SUM_K( 10, Type::real( -1456295425347.0 / 2642187500000.0 ), Type::real( 0.0 ), Type::real( 0.0 ), Type::real( 110740056.0 / 34765625.0 ), Type::real( -21221682384.0 / 4345703125.0 ), Type::real( 58859060169.0 / 13906250000.0 ), Type::real( -177381525069.0 / 1529687500000.0 ), Type::real( -28942485159.0 / 27812500000.0 ), Type::real( -1272297.0 / 312500000.0 ), Type::real( 5151.0 / 297616.0 ) );

                     CALCULATE_K( 11, Type::real( 0.84 ), buffer_wavefunction, buffer_reservoir );

                     INTERMEDIATE_SUM_K( 11, Type::real( 844300798.0 / 137013275.0 ), Type::real( 0.0 ), Type::real( 0.0 ), Type::real( -9997568.0 / 206035.0 ), Type::real( 49636624.0 / 1030175.0 ), Type::real( -3358834871.0 / 91067470.0 ), Type::real( -40456983.0 / 1813108.0 ), Type::real( 495817135.0 / 16647628.0 ), Type::real( -149375.0 / 84266.0 ), Type::real( 7470703125.0 / 1567431866.0 ), Type::real( 1562500000.0 / 72342361.0 ) );

                     CALCULATE_K( 12, Type::real( 1.0 ), buffer_wavefunction, buffer_reservoir );

                     INTERMEDIATE_SUM_K( 12, Type::real( -26225423.0 / 37371100.0 ), Type::real( 0.0 ), Type::real( 0.0 ), Type::real( 807744.0 / 98345.0 ), Type::real( -205584.0 / 37825.0 ), Type::real( 5882202.0 / 1278485.0 ), Type::real( -84543.0 / 432718.0 ), Type::real( -223415.0 / 39338.0 ), Type::real( -3625.0 / 6188.0 ), Type::real( 292968750.0 / 374084711.0 ), Type::real( 0.0 ), Type::real( 0.0 ) );

                     CALCULATE_K( 13, Type::real( 1.0 ), buffer_wavefunction, buffer_reservoir );

                     FINAL_SUM_K( 13, Type::real( 4241.0 / 88200.0 ), Type::real( 0.0 ), Type::real( 0.0 ), Type::real( 0.0 ), Type::real( 0.0 ), Type::real( 9946.0 / 23205.0 ), Type::real( -10449.0 / 1925.0 ), Type::real( 415449.0 / 1573075.0 ), Type::real( -2025.0 / 5096.0 ), Type::real( 48828125.0 / 44900856.0 ), Type::real( 48828125.0 / 9843561.0 ), Type::real( 463.0 / 12600.0 ), Type::real( 0.0 ) ); );
}

void PHOENIX::Solver::iterateVariableTimestepNSRK78() {
    bool accept = false;
    do {
        SOLVER_SEQUENCE( false /* no graph capture */,

                         CALCULATE_K( 1, Type::real( 0.0 ), wavefunction, reservoir );

                         INTERMEDIATE_SUM_K( 1, Type::real( 1.0 / 1000.0 ) );

                         CALCULATE_K( 2, Type::real( 0.001 ), buffer_wavefunction, buffer_reservoir );

                         INTERMEDIATE_SUM_K( 2, Type::real( -491.0 / 81.0 ), Type::real( 500.0 / 81.0 ) );

                         CALCULATE_K( 3, Type::real( 0.1111111111 ), buffer_wavefunction, buffer_reservoir );

                         INTERMEDIATE_SUM_K( 3, Type::real( 1.0 / 24.0 ), Type::real( 0.0 ), Type::real( 1.0 / 8.0 ) );

                         CALCULATE_K( 4, Type::real( 0.1666666667 ), buffer_wavefunction, buffer_reservoir );

                         INTERMEDIATE_SUM_K( 4, Type::real( 5.0 / 12.0 ), Type::real( 0.0 ), Type::real( -25.0 / 16.0 ), Type::real( 25.0 / 16.0 ) );

                         CALCULATE_K( 5, Type::real( 0.4166666667 ), buffer_wavefunction, buffer_reservoir );

                         INTERMEDIATE_SUM_K( 5, Type::real( 1.0 / 20.0 ), Type::real( 0.0 ), Type::real( 0.0 ), Type::real( 1.0 / 4.0 ), Type::real( 1.0 / 5.0 ) );

                         CALCULATE_K( 6, Type::real( 0.5 ), buffer_wavefunction, buffer_reservoir );

                         INTERMEDIATE_SUM_K( 6, Type::real( -43.0 / 180.0 ), Type::real( 0.0 ), Type::real( 0.0 ), Type::real( 127.0 / 108.0 ), Type::real( -329.0 / 135.0 ), Type::real( 7.0 / 3.0 ) );

                         CALCULATE_K( 7, Type::real( 0.8333333333 ), buffer_wavefunction, buffer_reservoir );

                         INTERMEDIATE_SUM_K( 7, Type::real( 27931.0 / 240300.0 ), Type::real( 0.0 ), Type::real( 0.0 ), Type::real( -631.0 / 16020.0 ), Type::real( 2459.0 / 6675.0 ), Type::real( -3572.0 / 12015.0 ), Type::real( 5.0 / 267.0 ) );

                         CALCULATE_K( 8, Type::real( 0.1666666667 ), buffer_wavefunction, buffer_reservoir );

                         INTERMEDIATE_SUM_K( 8, Type::real( 26114.0 / 12015.0 ), Type::real( 0.0 ), Type::real( 0.0 ), Type::real( -7480.0 / 801.0 ), Type::real( 67264.0 / 4005.0 ), Type::real( -30640.0 / 2403.0 ), Type::real( 1051.0 / 1335.0 ), Type::real( 3.0 ) );

                         CALCULATE_K( 9, Type::real( 0.6666666667 ), buffer_wavefunction, buffer_reservoir );

                         INTERMEDIATE_SUM_K( 9, Type::real( 33096587331.0 / 17382812500.0 ), Type::real( 0.0 ), Type::real( 0.0 ), Type::real( -1828977848.0 / 173828125.0 ), Type::real( 62801809904.0 / 4345703125.0 ), Type::real( -9389764774.0 / 869140625.0 ), Type::real( 6380757669.0 / 8691406250.0 ), Type::real( 98417891.0 / 19531250.0 ), Type::real( -1692691.0 / 39062500.0 ) );

                         CALCULATE_K( 10, Type::real( 0.76 ), buffer_wavefunction, buffer_reservoir );

                         INTERMEDIATE_SUM_K( 10, Type::real( -1456295425347.0 / 2642187500000.0 ), Type::real( 0.0 ), Type::real( 0.0 ), Type::real( 110740056.0 / 34765625.0 ), Type::real( -21221682384.0 / 4345703125.0 ), Type::real( 58859060169.0 / 13906250000.0 ), Type::real( -177381525069.0 / 1529687500000.0 ), Type::real( -28942485159.0 / 27812500000.0 ), Type::real( -1272297.0 / 312500000.0 ), Type::real( 5151.0 / 297616.0 ) );

                         CALCULATE_K( 11, Type::real( 0.84 ), buffer_wavefunction, buffer_reservoir );

                         INTERMEDIATE_SUM_K( 11, Type::real( 844300798.0 / 137013275.0 ), Type::real( 0.0 ), Type::real( 0.0 ), Type::real( -9997568.0 / 206035.0 ), Type::real( 49636624.0 / 1030175.0 ), Type::real( -3358834871.0 / 91067470.0 ), Type::real( -40456983.0 / 1813108.0 ), Type::real( 495817135.0 / 16647628.0 ), Type::real( -149375.0 / 84266.0 ), Type::real( 7470703125.0 / 1567431866.0 ), Type::real( 1562500000.0 / 72342361.0 ) );

                         CALCULATE_K( 12, Type::real( 1.0 ), buffer_wavefunction, buffer_reservoir );

                         INTERMEDIATE_SUM_K( 12, Type::real( -26225423.0 / 37371100.0 ), Type::real( 0.0 ), Type::real( 0.0 ), Type::real( 807744.0 / 98345.0 ), Type::real( -205584.0 / 37825.0 ), Type::real( 5882202.0 / 1278485.0 ), Type::real( -84543.0 / 432718.0 ), Type::real( -223415.0 / 39338.0 ), Type::real( -3625.0 / 6188.0 ), Type::real( 292968750.0 / 374084711.0 ), Type::real( 0.0 ), Type::real( 0.0 ) );

                         CALCULATE_K( 13, Type::real( 1.0 ), buffer_wavefunction, buffer_reservoir );

                         INTERMEDIATE_SUM_K( 13, Type::real( 4241.0 / 88200.0 ), Type::real( 0.0 ), Type::real( 0.0 ), Type::real( 0.0 ), Type::real( 0.0 ), Type::real( 9946.0 / 23205.0 ), Type::real( -10449.0 / 1925.0 ), Type::real( 415449.0 / 1573075.0 ), Type::real( -2025.0 / 5096.0 ), Type::real( 48828125.0 / 44900856.0 ), Type::real( 48828125.0 / 9843561.0 ), Type::real( 463.0 / 12600.0 ), Type::real( 0.0 ) );

                         ERROR_K( 13, Type::real( 4241.0 / 88200.0 - 3799.0 / 79800.0 ), Type::real( 0.0 ), Type::real( 0.0 ), Type::real( 0.0 ), Type::real( 0.0 ), Type::real( 9946.0 / 23205.0 - 538.0 / 1365.0 ), Type::real( -10449.0 / 1925.0 - 351.0 / 1925.0 ), Type::real( 415449.0 / 1573075.0 - 4149.0 / 15575.0 ), Type::real( -2025.0 / 5096.0 - ( -45.0 / 392.0 ) ), Type::real( 48828125.0 / 44900856.0 - 48828125.0 / 284372088.0 ), Type::real( 48828125.0 / 9843561.0 - 0.0 ),
                                  Type::real( 463.0 / 12600.0 - 0.0 ), Type::real( 0.0 - 221.0 / 4200.0 ) );

        );

        auto msum = matrix.buffer_wavefunction_plus.transformReduce( Type::complex( 0.0 ), CUDAMatrix<Type::complex>::transform_abs2(), CUDAMatrix<Type::complex>::transform_sum() );
        Type::real normalization_factor = CUDA::real( msum );
        Type::real integrated_error = matrix.rk_error.sum();
        Type::real final_error = std::abs( integrated_error / normalization_factor );

        Type::real dh_arg = system.tolerance / 2.0 / CUDA::max( std::numeric_limits<Type::real>::min(), final_error );
        Type::real dh = std::pow( dh_arg, Type::real( 1.0 / 8.0 ) );

        if ( std::isnan( dh ) || std::isnan( final_error ) ) {
            dh = Type::real( 0.9 );
            final_error = std::numeric_limits<Type::real>::max();
        }

        Type::real new_dt = std::min( system.p.dt * dh, system.dt_max );
        if ( new_dt < system.dt_min ) {
            new_dt = system.dt_min;
            accept = true;
        }
        system.p.dt = new_dt;
        updateKernelTime();

        if ( final_error < system.tolerance ) {
            accept = true;
            matrix.wavefunction_plus.swap( matrix.buffer_wavefunction_plus );
            matrix.reservoir_plus.swap( matrix.buffer_reservoir_plus );
            if ( system.use_twin_mode ) {
                matrix.wavefunction_minus.swap( matrix.buffer_wavefunction_minus );
                matrix.reservoir_minus.swap( matrix.buffer_reservoir_minus );
            }
        }
    } while ( !accept );
}
