#include "hip/hip_runtime.h"
#include <omp.h>

// Include Cuda Kernel headers
#include "cuda/typedef.cuh"
#include "kernel/kernel_compute.cuh"
#include "kernel/kernel_summation.cuh"
#include "kernel/kernel_halo.cuh"
#include "system/system_parameters.hpp"
#include "cuda/cuda_matrix.cuh"
#include "solver/gpu_solver.hpp"
#include "misc/commandline_io.hpp"

void PHOENIX::Solver::iterateFixedTimestepFehlberg2() {
    SOLVER_SEQUENCE( true /*Capture CUDA Graph*/,

                     CALCULATE_K( 1, Type::real( 0.0 ), wavefunction, reservoir );

                     INTERMEDIATE_SUM_K( 1, Type::real( 1.0 / 2.0 ) );

                     CALCULATE_K( 2, Type::real( 1.0 / 2.0 ), buffer_wavefunction, buffer_reservoir );

                     INTERMEDIATE_SUM_K( 2, Type::real( 1.0 / 256.0 ), Type::real( 255.0 / 256.0 ) );

                     CALCULATE_K( 3, Type::real( 1.0 ), buffer_wavefunction, buffer_reservoir );

                     FINAL_SUM_K( 3, Type::real( 1.0 / 512.0 ), Type::real( 255.0 / 256.0 ), Type::real( 1.0 / 512.0 ) );

    );
}

void PHOENIX::Solver::iterateVariableTimestepFehlberg2() {
    bool accept = false;
    do {
        SOLVER_SEQUENCE( false /*Capture CUDA Graph*/,

                         CALCULATE_K( 1, Type::real( 0.0 ), wavefunction, reservoir );

                         INTERMEDIATE_SUM_K( 1, Type::real( 1.0 / 2.0 ) );

                         CALCULATE_K( 2, Type::real( 1.0 / 2.0 ), buffer_wavefunction, buffer_reservoir );

                         INTERMEDIATE_SUM_K( 2, Type::real( 1.0 / 256.0 ), Type::real( 255.0 / 256.0 ) );

                         CALCULATE_K( 3, Type::real( 1.0 ), buffer_wavefunction, buffer_reservoir );

                         INTERMEDIATE_SUM_K( 3, Type::real( 1.0 / 512.0 ), Type::real( 255.0 / 256.0 ), Type::real( 1.0 / 512.0 ) );

                         ERROR_K( 3, Type::real( 1.0 / 512.0 - 1.0 / 256.0 ), Type::real( 0.0 ), Type::real( 1.0 / 512.0 ) ); );

        auto msum = matrix.buffer_wavefunction_plus.transformReduce( Type::complex( 0.0 ), CUDAMatrix<Type::complex>::transform_abs2(), CUDAMatrix<Type::complex>::transform_sum() );
        Type::real normalization_factor = CUDA::sqrt( CUDA::real( msum ) );
        Type::real integrated_error = std::sqrt( matrix.rk_error.sum() );

        Type::real final_error = std::abs( integrated_error / normalization_factor );
        Type::real dh_arg = system.tolerance / 2.0 / CUDA::max( std::numeric_limits<Type::real>::min(), final_error );
        Type::real dh = std::pow<Type::real>( dh_arg, Type::real( 0.5 ) );

        if ( std::isnan( dh ) ) {
            dh = 0.9;
            final_error = std::numeric_limits<Type::real>::max();
        }
        if ( std::isnan( final_error ) ) {
            dh = 0.9;
            final_error = std::numeric_limits<Type::real>::max();
        }

        //  Set new timestep
        Type::real new_dt = std::min( system.p.dt * dh, system.dt_max );
        if ( new_dt < system.dt_min ) {
            new_dt = system.dt_min;
            accept = true;
        }
        system.p.dt = new_dt;

        updateKernelTime();

        if ( final_error < system.tolerance ) {
            accept = true;
            matrix.wavefunction_plus.swap( matrix.buffer_wavefunction_plus );
            matrix.reservoir_plus.swap( matrix.buffer_reservoir_plus );
            if ( system.use_twin_mode ) {
                matrix.wavefunction_minus.swap( matrix.buffer_wavefunction_minus );
                matrix.reservoir_minus.swap( matrix.buffer_reservoir_minus );
            }
        }
    } while ( !accept );
}

void PHOENIX::Solver::iterateFixedTimestepFehlberg5() {
    SOLVER_SEQUENCE( true /*Capture CUDA Graph*/,

                     CALCULATE_K( 1, Type::real( 0.0 ), wavefunction, reservoir );

                     INTERMEDIATE_SUM_K( 1, Type::real( 1.0 / 40.0 ) );

                     CALCULATE_K( 2, Type::real( 1.0 / 4.0 ), buffer_wavefunction, buffer_reservoir );

                     INTERMEDIATE_SUM_K( 2, Type::real( 3.0 / 32.0 ), Type::real( 9.0 / 32.0 ) );

                     CALCULATE_K( 3, Type::real( 3.0 / 8.0 ), buffer_wavefunction, buffer_reservoir );

                     INTERMEDIATE_SUM_K( 3, Type::real( 1932.0 / 2197.0 ), Type::real( -7200.0 / 2197.0 ), Type::real( 7296.0 / 2197.0 ) );

                     CALCULATE_K( 4, Type::real( 12.0 / 13.0 ), buffer_wavefunction, buffer_reservoir );

                     INTERMEDIATE_SUM_K( 4, Type::real( 439.0 / 216.0 ), Type::real( -8.0 ), Type::real( 3680.0 / 513.0 ), Type::real( -845.0 / 4104.0 ) );

                     CALCULATE_K( 5, Type::real( 1.0 ), buffer_wavefunction, buffer_reservoir );

                     INTERMEDIATE_SUM_K( 5, Type::real( -8.0 / 27.0 ), Type::real( 2.0 ), Type::real( -3544.0 / 2565.0 ), Type::real( 1859.0 / 4104.0 ), Type::real( -11.0 / 40.0 ) );

                     CALCULATE_K( 6, Type::real( 1.0 / 2.0 ), buffer_wavefunction, buffer_reservoir );

                     FINAL_SUM_K( 6, Type::real( 16.0 / 135.0 ), Type::real( 0.0 ), Type::real( 6656.0 / 12825.0 ), Type::real( 28561.0 / 56430.0 ), Type::real( -9.0 / 50.0 ), Type::real( 2.0 / 55.0 ) );

    );
}

void PHOENIX::Solver::iteratevariableTimestepFehlberg5() {
    bool accept = false;
    do {
        SOLVER_SEQUENCE( false /*Capture CUDA Graph*/,

                         CALCULATE_K( 1, Type::real( 0.0 ), wavefunction, reservoir );

                         INTERMEDIATE_SUM_K( 1, Type::real( 1.0 / 40.0 ) );

                         CALCULATE_K( 2, Type::real( 1.0 / 4.0 ), buffer_wavefunction, buffer_reservoir );

                         INTERMEDIATE_SUM_K( 2, Type::real( 3.0 / 32.0 ), Type::real( 9.0 / 32.0 ) );

                         CALCULATE_K( 3, Type::real( 3.0 / 8.0 ), buffer_wavefunction, buffer_reservoir );

                         INTERMEDIATE_SUM_K( 3, Type::real( 1932.0 / 2197.0 ), Type::real( -7200.0 / 2197.0 ), Type::real( 7296.0 / 2197.0 ) );

                         CALCULATE_K( 4, Type::real( 12.0 / 13.0 ), buffer_wavefunction, buffer_reservoir );

                         INTERMEDIATE_SUM_K( 4, Type::real( 439.0 / 216.0 ), Type::real( -8.0 ), Type::real( 3680.0 / 513.0 ), Type::real( -845.0 / 4104.0 ) );

                         CALCULATE_K( 5, Type::real( 1.0 ), buffer_wavefunction, buffer_reservoir );

                         INTERMEDIATE_SUM_K( 5, Type::real( -8.0 / 27.0 ), Type::real( 2.0 ), Type::real( -3544.0 / 2565.0 ), Type::real( 1859.0 / 4104.0 ), Type::real( -11.0 / 40.0 ) );

                         CALCULATE_K( 6, Type::real( 1.0 / 2.0 ), buffer_wavefunction, buffer_reservoir );

                         // Write result to buffer_ instead of wavefunction_
                         INTERMEDIATE_SUM_K( 6, Type::real( 16.0 / 135.0 ), Type::real( 0.0 ), Type::real( 6656.0 / 12825.0 ), Type::real( 28561.0 / 56430.0 ), Type::real( -9.0 / 50.0 ), Type::real( 2.0 / 55.0 ) );
                         //FINAL_SUM_K( 6, Type::real( 16.0 / 135.0 ), Type::real( 0.0 ), Type::real( 6656.0 / 12825.0 ), Type::real( 28561.0 / 56430.0 ), Type::real( -9.0 / 50.0 ), Type::real( 2.0 / 55.0 ) );

                         // Calculate the error. If the error is small enough, accept the step and move buffer_ into wavefunction_.
                         ERROR_K( 6, Type::real( 16.0 / 135.0 - 25.0 / 216.0 ), Type::real( 0.0 ), Type::real( 6656.0 / 12825.0 - 1408.0 / 2565.0 ), Type::real( 28561.0 / 56430.0 - 2197.0 / 4104.0 ), Type::real( -9.0 / 50.0 - 1.0 / 5.0 ), Type::real( 2.0 / 55.0 ) );

        );

        //auto [min, max] = matrix.buffer_wavefunction_plus.extrema();
        auto msum = matrix.buffer_wavefunction_plus.transformReduce( Type::complex( 0.0 ), CUDAMatrix<Type::complex>::transform_abs2(), CUDAMatrix<Type::complex>::transform_sum() );
        Type::real normalization_factor = CUDA::sqrt( CUDA::real( msum ) );
        Type::real integrated_error = std::sqrt( matrix.rk_error.sum() );

        Type::real final_error = std::abs( integrated_error / normalization_factor );
        Type::real dh_arg = system.tolerance / 2.0 / CUDA::max( std::numeric_limits<Type::real>::min(), final_error );
        Type::real dh = std::pow<Type::real>( dh_arg, Type::real( 0.25 ) );

        if ( std::isnan( dh ) ) {
            dh = 0.9;
            final_error = std::numeric_limits<Type::real>::max();
        }
        if ( std::isnan( final_error ) ) {
            dh = 0.9;
            final_error = std::numeric_limits<Type::real>::max();
        }

        //  Set new timestep
        Type::real new_dt = std::min( system.p.dt * dh, system.dt_max );
        //if ( dh < 1.0 )
        //new_dt = std::max( system.p.dt - system.dt_min * std::floor( 1.0 / dh ), system.dt_min );
        //else
        //new_dt = std::min( system.p.dt + system.dt_min * std::floor( dh ), system.dt_max );
        if ( new_dt < system.dt_min ) {
            new_dt = system.dt_min;
            accept = true;
        }
        system.p.dt = new_dt;

        updateKernelTime();

        if ( final_error < system.tolerance ) {
            accept = true;
            matrix.wavefunction_plus.swap( matrix.buffer_wavefunction_plus );
            matrix.reservoir_plus.swap( matrix.buffer_reservoir_plus );
            if ( system.use_twin_mode ) {
                matrix.wavefunction_minus.swap( matrix.buffer_wavefunction_minus );
                matrix.reservoir_minus.swap( matrix.buffer_reservoir_minus );
            }
            //std::cout << "ACCEPTED " << final_error << ", norm = " << normalization_factor << ", dh = " << dh << " --> new dt = " << new_dt << " (old: " << system.p.dt << ")" << std::endl;
        }

        //std::cout << final_error << ", norm = " << normalization_factor << ", dh = " << dh << " --> new dt = " << new_dt << " (old: " << system.p.dt << ")" << std::endl;
    } while ( !accept );
}