#include <omp.h>

// Include Cuda Kernel headers
#include "cuda/typedef.cuh"
#include "kernel/kernel_compute.cuh"
#include "kernel/kernel_summation.cuh"
#include "kernel/kernel_halo.cuh"
#include "system/system_parameters.hpp"
#include "cuda/cuda_matrix.cuh"
#include "solver/gpu_solver.hpp"
#include "misc/commandline_io.hpp"

/*
 * This function iterates the Runge Kutta Kernel using a fixed time step.
 * A 4th order Runge-Kutta method is used. This function calls a single
 * rungeFuncSum function with varying delta-t. Calculation of the inputs
 * for the next rungeFuncKernel call is done in the rungeFuncSum function.
 * The general implementation of the RK4 method goes as follows:
 * ------------------------------------------------------------------------------
 * k1 = f(t, y) = rungeFuncKernel(current)
 * input_for_k2 = current + 0.5 * dt * k1
 * k2 = f(t + 0.5 * dt, input_for_k2) = rungeFuncKernel(input_for_k2)
 * input_for_k3 = current + 0.5 * dt * k2
 * k3 = f(t + 0.5 * dt, input_for_k3) = rungeFuncKernel(input_for_k3)
 * input_for_k4 = current + dt * k3
 * k4 = f(t + dt, input_for_k4) = rungeFuncKernel(input_for_k4) 
 * next = current + dt * (1/6 * k1 + 1/3 * k2 + 1/3 * k3 + 1/6 * k4)
 * ------------------------------------------------------------------------------ 
 * The Runge method iterates psi,k1-k4 to psi_next using a wave-like approach.
 * We calculate 4 rows of k1, 3 rows of k2, 2 rows of k3 and 1 row of k4 before the first iteration.
 * Then, we iterate all of the remaining rows after each other, incrementing the buffer for the next iteration.
 */

void PC3::Solver::iterateFixedTimestepRungeKutta4() {   
 
    SOLVER_SEQUENCE( true /*Capture CUDA Graph*/,  
   
                     CALCULATE_K( 1, wavefunction, reservoir ); 
 
                     INTERMEDIATE_SUM_K( 1, 0.5f );   
    
                     CALCULATE_K( 2, buffer_wavefunction, buffer_reservoir );

                     INTERMEDIATE_SUM_K( 2, 0.5f );

                     CALCULATE_K( 3, buffer_wavefunction, buffer_reservoir ); 

                     INTERMEDIATE_SUM_K( 3, 1.0f );

                     CALCULATE_K( 4, buffer_wavefunction, buffer_reservoir );  

                     FINAL_SUM_K( 4, 1.01f / 6.0f, 1.0f / 3.0f, 1.0f / 3.0f, 1.0f / 6.0f ); 

    );

}