#include "hip/hip_runtime.h"
#include <vector>
#include <string>

#include "cuda/typedef.cuh"
#include "solver/gpu_solver.hpp"

void PC3::Solver::cacheValues() {
    // System Time
    cache_map_scalar["t"].emplace_back( system.p.t );

    // Min and Max
    auto [min_plus, max_plus] = matrix.wavefunction_plus.extrema();
    cache_map_scalar["min_plus"].emplace_back( CUDA::abs( min_plus ) );
    cache_map_scalar["max_plus"].emplace_back( CUDA::abs( max_plus ) );

    // Output Pulse, Pump and Potential Envelope functions to cache_map_scalar
    for ( int g = 0; g < system.pulse.groupSize(); g++ ) {
        if ( system.pulse.temporal[g] & PC3::Envelope::Temporal::Constant )
            continue;
        Type::complex pulse = system.pulse.temporal_envelope[g];
        cache_map_scalar["pulse_" + std::to_string( g ) + "_real"].push_back( PC3::CUDA::real( pulse ) );
        cache_map_scalar["pulse_" + std::to_string( g ) + "_imag"].push_back( PC3::CUDA::imag( pulse ) );
    }
    for ( int g = 0; g < system.pump.groupSize(); g++ ) {
        if ( system.pump.temporal[g] & PC3::Envelope::Temporal::Constant )
            continue;
        Type::real pump = PC3::CUDA::real( system.pump.temporal_envelope[g] );
        cache_map_scalar["pump_" + std::to_string( g )].push_back( PC3::CUDA::real( pump ) );
    }
    for ( int g = 0; g < system.potential.groupSize(); g++ ) {
        if ( system.potential.temporal[g] & PC3::Envelope::Temporal::Constant )
            continue;
        Type::real potential = PC3::CUDA::real( system.potential.temporal_envelope[g] );
        cache_map_scalar["potential_" + std::to_string( g )].push_back( PC3::CUDA::real( potential ) );
    }

    // TE/TM Guard
    if ( not system.p.use_twin_mode )
        return;

    // Same for _minus component if use_twin_mode is true
    const auto [min_minus, max_minus] = matrix.wavefunction_minus.extrema();
    cache_map_scalar["min_minus"].emplace_back( CUDA::abs( min_minus ) );
    cache_map_scalar["max_minus"].emplace_back( CUDA::abs( max_minus ) );
}

void PC3::Solver::cacheToFiles() {
    if ( not system.doOutput( "all", "max", "scalar" ) )
        return;

    auto& file_max = filehandler.getFile( "scalar" );
    file_max << "index ";
    for ( const auto& [key, _] : cache_map_scalar ) file_max << key << " ";
    file_max << "\n";
    for ( int i = 0; i < cache_map_scalar["t"].size(); i++ ) {
        file_max << i << " ";
        for ( const auto& [_, vec] : cache_map_scalar ) file_max << vec[i] << " ";
        file_max << "\n";
    }
    file_max.close();
}

// TODO: Support Multiple History Outputs, and also support piping them into a single file.
// something like "append" mode, that doesnt open a new file but instead appends to the existing one.
PC3::Type::uint32 _local_history_output_counter = 1; // output_history_matrix_every
void PC3::Solver::cacheMatrices() {
    if ( not system.do_output_history_matrix ) // Don't output history matrix
        return;
    if ( system.p.t < system.output_history_start_time ) // Start time not reached
        return;
    if ( _local_history_output_counter < system.output_history_matrix_every ) { // Not yet time to output
        _local_history_output_counter++;
        return;
    }
    std::string suffix = "_" + std::to_string( system.p.t );
    _local_history_output_counter = 1;
    outputMatrices( system.history_matrix_start_x, system.history_matrix_end_x, system.history_matrix_start_y, system.history_matrix_end_y, system.history_matrix_output_increment,
                    suffix, "timeoutput/" );
}
